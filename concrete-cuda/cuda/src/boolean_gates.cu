#ifndef CUDA_BOOLEAN_GATES_CU
#define CUDA_BOOLEAN_GATES_CU

#include "bootstrap.h"
#include "device.h"
#include "hip/hip_runtime_api.h"
#include "keyswitch.h"
#include "linear_algebra.h"

constexpr int PLAINTEXT_TRUE{1 << (32 - 3)};
constexpr int PLAINTEXT_FALSE{7 << (32 - 3)};

extern "C" void cuda_boolean_not_32(void *v_stream, uint32_t gpu_index,
                                    void *lwe_array_out, void *lwe_array_in,
                                    uint32_t input_lwe_dimension,
                                    uint32_t input_lwe_ciphertext_count) {

  cuda_negate_lwe_ciphertext_vector_32(v_stream, gpu_index, lwe_array_out,
                                       lwe_array_in, input_lwe_dimension,
                                       input_lwe_ciphertext_count);
}

extern "C" void cuda_boolean_and_32(
    void *v_stream, uint32_t gpu_index, void *lwe_array_out,
    void *lwe_array_in_1, void *lwe_array_in_2, void *bootstrapping_key,
    void *ksk, uint32_t input_lwe_dimension, uint32_t glwe_dimension,
    uint32_t polynomial_size, uint32_t pbs_base_log, uint32_t pbs_level_count,
    uint32_t ks_base_log, uint32_t ks_level_count,
    uint32_t input_lwe_ciphertext_count, uint32_t max_shared_memory) {

  auto stream = static_cast<hipStream_t *>(v_stream);

  uint32_t *lwe_buffer = (uint32_t *)cuda_malloc_async(
      (input_lwe_dimension + 1) * input_lwe_ciphertext_count * sizeof(uint32_t),
      stream, gpu_index);
  // 1. Add the two ciphertexts
  cuda_add_lwe_ciphertext_vector_32(
      v_stream, gpu_index, lwe_buffer, lwe_array_in_1, lwe_array_in_2,
      input_lwe_dimension, input_lwe_ciphertext_count);
  // 2. Add "false" plaintext, where "false" is 7 << (32 - 3)
  uint32_t *h_false_plaintext_array =
      (uint32_t *)malloc(input_lwe_ciphertext_count * sizeof(uint32_t));
  for (uint index = 0; index < input_lwe_ciphertext_count; index++) {
    h_false_plaintext_array[index] = PLAINTEXT_FALSE;
  }
  uint32_t *false_plaintext_array = (uint32_t *)cuda_malloc_async(
      input_lwe_ciphertext_count * sizeof(uint32_t), stream, gpu_index);
  cuda_memcpy_async_to_gpu(false_plaintext_array, h_false_plaintext_array,
                           input_lwe_ciphertext_count * sizeof(uint32_t),
                           stream, gpu_index);
  checkCudaErrors(hipGetLastError());

  uint32_t *lwe_buffer_2 = (uint32_t *)cuda_malloc_async(
      (input_lwe_dimension + 1) * input_lwe_ciphertext_count * sizeof(uint32_t),
      stream, gpu_index);
  cuda_add_lwe_ciphertext_vector_plaintext_vector_32(
      v_stream, gpu_index, lwe_buffer_2, lwe_buffer, false_plaintext_array,
      input_lwe_dimension, input_lwe_ciphertext_count);

  cuda_drop_async(lwe_buffer, stream, gpu_index);
  cuda_drop_async(false_plaintext_array, stream, gpu_index);
  free(h_false_plaintext_array);

  // 3. Compute a PBS with the LUT created below
  uint32_t *h_pbs_lut = (uint32_t *)malloc((glwe_dimension + 1) *
                                           polynomial_size * sizeof(uint32_t));
  for (uint index = 0; index < (glwe_dimension + 1) * polynomial_size;
       index++) {
    h_pbs_lut[index] =
        index < (glwe_dimension * polynomial_size) ? 0 : PLAINTEXT_TRUE;
  }
  uint32_t *pbs_lut = (uint32_t *)cuda_malloc_async(
      (glwe_dimension + 1) * polynomial_size * sizeof(uint32_t), stream,
      gpu_index);
  cuda_memcpy_async_to_gpu(pbs_lut, h_pbs_lut,
                           (glwe_dimension + 1) * polynomial_size *
                               sizeof(uint32_t),
                           stream, gpu_index);
  checkCudaErrors(hipGetLastError());
  uint32_t *h_pbs_lut_indexes =
      (uint32_t *)malloc(input_lwe_ciphertext_count * sizeof(uint32_t));
  for (uint index = 0; index < input_lwe_ciphertext_count; index++) {
    h_pbs_lut_indexes[index] = 0;
  }
  uint32_t *pbs_lut_indexes = (uint32_t *)cuda_malloc_async(
      input_lwe_ciphertext_count * sizeof(uint32_t), stream, gpu_index);
  cuda_memcpy_async_to_gpu(pbs_lut_indexes, h_pbs_lut_indexes,
                           input_lwe_ciphertext_count * sizeof(uint32_t),
                           stream, gpu_index);
  checkCudaErrors(hipGetLastError());
  uint32_t *lwe_pbs_buffer = (uint32_t *)cuda_malloc_async(
      (glwe_dimension * polynomial_size + 1) * input_lwe_ciphertext_count *
          sizeof(uint32_t),
      stream, gpu_index);
  checkCudaErrors(hipGetLastError());

  cuda_bootstrap_amortized_lwe_ciphertext_vector_32(
      v_stream, gpu_index, lwe_pbs_buffer, pbs_lut, pbs_lut_indexes,
      lwe_buffer_2, bootstrapping_key, input_lwe_dimension, glwe_dimension,
      polynomial_size, pbs_base_log, pbs_level_count,
      input_lwe_ciphertext_count, 1, 0, max_shared_memory);
  checkCudaErrors(hipGetLastError());

  cuda_drop_async(lwe_buffer_2, stream, gpu_index);
  cuda_drop_async(pbs_lut, stream, gpu_index);
  cuda_drop_async(pbs_lut_indexes, stream, gpu_index);
  free(h_pbs_lut);
  free(h_pbs_lut_indexes);

  cuda_keyswitch_lwe_ciphertext_vector_32(
      v_stream, gpu_index, lwe_array_out, lwe_pbs_buffer, ksk,
      glwe_dimension * polynomial_size, input_lwe_dimension, ks_base_log,
      ks_level_count, input_lwe_ciphertext_count);

  cuda_drop_async(lwe_pbs_buffer, stream, gpu_index);
}
#endif // CUDA_BOOLEAN_GATES_CU
