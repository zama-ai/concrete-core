#include "hip/hip_runtime.h"
#include "bootstrap_low_latency.cuh"

/* Perform bootstrapping on a batch of input LWE ciphertexts
 *
 *  - lwe_out: output batch of num_samples bootstrapped ciphertexts c =
 * (a0,..an-1,b) where n is the LWE dimension
 *  - lut_vector: should hold as many test vectors of size polynomial_size
 * as there are input ciphertexts, but actually holds
 * num_lut_vectors vectors to reduce memory usage
 *  - lut_vector_indexes: stores the index corresponding to
 * which test vector to use for each sample in
 * lut_vector
 *  - lwe_in: input batch of num_samples LWE ciphertexts, containing n
 * mask values + 1 body value
 *  - bootstrapping_key: RGSW encryption of the LWE secret key sk1
 * under secret key sk2
 * bsk = Z + sk1 H
 * where H is the gadget matrix and Z is a matrix (k+1).l
 * containing GLWE encryptions of 0 under sk2.
 * bsk is thus a tensor of size (k+1)^2.l.N.n
 * where l is the number of decomposition levels and
 * k is the GLWE dimension, N is the polynomial size for
 * GLWE. The polynomial size for GLWE and the test vector
 * are the same because they have to be in the same ring
 * to be multiplied.
 * Note: it is necessary to generate (k+1).k.l.N.n
 * uniformly random coefficients for the zero encryptions
 * - lwe_dimension: size of the Torus vector used to encrypt the input
 * LWE ciphertexts - referred to as n above (~ 600)
 * - polynomial_size: size of the test polynomial (test vector) and size of the
 * GLWE polynomial (~1024)
 * - base_log: log base used for the gadget matrix - B = 2^base_log (~8)
 * - l_gadget: number of decomposition levels in the gadget matrix (~4)
 * - num_samples: number of encrypted input messages
 * - num_lut_vectors: parameter to set the actual number of test vectors to be
 * used
 * - q: number of bytes in the integer representation (32 or 64)
 *
 * This function calls a wrapper to a device kernel that performs the
 * bootstrapping:
 * 	- the kernel is templatized based on integer discretization and
 * polynomial degree
 * 	- num_samples blocks of threads are launched, where each thread is going
 * to handle one or more polynomial coefficients at each stage:
 * 		- perform the blind rotation
 * 		- round the result
 * 		- decompose into l_gadget levels, then for each level:
 * 		  - switch to the FFT domain
 * 		  - multiply with the bootstrapping key
 * 		  - come back to the coefficients representation
 * 	- between each stage a synchronization of the threads is necessary
 * 	- in case the device has enough shared memory, temporary arrays used for
 * the different stages (accumulators) are stored into the shared memory
 * 	- the accumulators serve to combine the results for all decomposition
 * levels
 * 	- the constant memory (64K) is used for storing the roots of identity
 * values for the FFT
 */
void cuda_bootstrap_low_latency_lwe_ciphertext_vector_32(
        void *v_stream,
        void *lwe_out,
        void *lut_vector,
        void *lut_vector_indexes,
        void *lwe_in,
        void *bootstrapping_key,
        uint32_t lwe_dimension,
        uint32_t glwe_dimension,
        uint32_t polynomial_size,
        uint32_t base_log,
        uint32_t l_gadget,
        uint32_t num_samples,
        uint32_t num_lut_vectors,
        uint32_t lwe_idx,
        uint32_t max_shared_memory) {

    assert(("Error (GPU low latency PBS): base log should be <= 16", base_log <= 16));
    assert(("Error (GPU low latency PBS): glwe_dimension should be equal to 1",
            glwe_dimension == 1));
    assert(("Error (GPU low latency PBS): polynomial size should be one of 512, 1024, 2048",
            polynomial_size == 512 || polynomial_size == 1024 || polynomial_size == 2048));
    // The number of samples should be lower than SM/(4 * (k + 1) * l) (the
    // factor 4 being related to the occupancy of 50%). The only supported
    // value for k is 1, so k + 1 = 2 for now.
    int number_of_sm = 0;
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, 0);
    assert(("Error (GPU low latency PBS): the number of input LWEs must be lower or equal to the "
            "number of streaming multiprocessors on the device divided by 8 * l_gadget",
            num_samples <= number_of_sm / 4. / 2. / l_gadget));

  switch (polynomial_size) {
  case 512:
    host_bootstrap_low_latency<uint32_t, Degree<512>>(
        v_stream, (uint32_t *)lwe_out, (uint32_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint32_t *)lwe_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size,
        base_log, l_gadget, num_samples,
        num_lut_vectors);
    break;
  case 1024:
    host_bootstrap_low_latency<uint32_t, Degree<1024>>(
        v_stream, (uint32_t *)lwe_out, (uint32_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint32_t *)lwe_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size,
        base_log, l_gadget, num_samples,
        num_lut_vectors);
    break;
  case 2048:
    host_bootstrap_low_latency<uint32_t, Degree<2048>>(
        v_stream, (uint32_t *)lwe_out, (uint32_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint32_t *)lwe_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size,
        base_log, l_gadget, num_samples,
        num_lut_vectors);
    break;
  case 4096:
    host_bootstrap_low_latency<uint32_t, Degree<4096>>(
        v_stream, (uint32_t *)lwe_out, (uint32_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint32_t *)lwe_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size,
        base_log, l_gadget, num_samples,
        num_lut_vectors);
    break;
  case 8192:
    host_bootstrap_low_latency<uint32_t, Degree<8192>>(
        v_stream, (uint32_t *)lwe_out, (uint32_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint32_t *)lwe_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size,
        base_log, l_gadget, num_samples,
        num_lut_vectors);
    break;
  default:
    break;
  }
}

void cuda_bootstrap_low_latency_lwe_ciphertext_vector_64(
        void *v_stream,
        void *lwe_out,
        void *lut_vector,
        void *lut_vector_indexes,
        void *lwe_in,
        void *bootstrapping_key,
        uint32_t lwe_dimension,
        uint32_t glwe_dimension,
        uint32_t polynomial_size,
        uint32_t base_log,
        uint32_t l_gadget,
        uint32_t num_samples,
        uint32_t num_lut_vectors,
        uint32_t lwe_idx,
        uint32_t max_shared_memory) {

    assert(("Error (GPU low latency PBS): base log should be <= 16", base_log <= 16));
    assert(("Error (GPU low latency PBS): glwe_dimension should be equal to 1", 
            glwe_dimension == 1));
    assert(("Error (GPU low latency PBS): polynomial size should be one of 512, 1024, 2048",
            polynomial_size == 512 || polynomial_size == 1024 || polynomial_size == 2048));
    // The number of samples should be lower than SM/(4 * (k + 1) * l) (the
    // factor 4 being related to the occupancy of 50%). The only supported
    // value for k is 1, so k + 1 = 2 for now.
    int number_of_sm = 0;
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, 0);
    assert(("Error (GPU low latency PBS): the number of input LWEs must be lower or equal to the "
            "number of streaming multiprocessors on the device divided by 8 * l_gadget",
            num_samples <= number_of_sm / 4. / 2. / l_gadget));

  switch (polynomial_size) {
  case 512:
    host_bootstrap_low_latency<uint64_t, Degree<512>>(
        v_stream, (uint64_t *)lwe_out, (uint64_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint64_t *)lwe_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size,
        base_log, l_gadget, num_samples,
        num_lut_vectors);
    break;
  case 1024:
    host_bootstrap_low_latency<uint64_t, Degree<1024>>(
        v_stream, (uint64_t *)lwe_out, (uint64_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint64_t *)lwe_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size,
        base_log, l_gadget, num_samples,
        num_lut_vectors);
    break;
  case 2048:
    host_bootstrap_low_latency<uint64_t, Degree<2048>>(
        v_stream, (uint64_t *)lwe_out, (uint64_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint64_t *)lwe_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size,
        base_log, l_gadget, num_samples,
        num_lut_vectors);
    break;
  case 4096:
    host_bootstrap_low_latency<uint64_t, Degree<4096>>(
        v_stream, (uint64_t *)lwe_out, (uint64_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint64_t *)lwe_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size,
        base_log, l_gadget, num_samples,
        num_lut_vectors);
    break;
  case 8192:
    host_bootstrap_low_latency<uint64_t, Degree<8192>>(
        v_stream, (uint64_t *)lwe_out, (uint64_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint64_t *)lwe_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size,
        base_log, l_gadget, num_samples,
        num_lut_vectors);
    break;
  default:
    break;
  }
}

