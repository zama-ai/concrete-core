#include "hip/hip_runtime.h"
#include "bit_extraction.cuh"

void cuda_extract_bits_32(
    void *v_stream, uint32_t gpu_index, void *list_lwe_array_out,
    void *lwe_array_in, void *lwe_array_in_buffer,
    void *lwe_array_in_shifted_buffer, void *lwe_array_out_ks_buffer,
    void *lwe_array_out_pbs_buffer, void *lut_pbs, void *lut_vector_indexes,
    void *ksk, void *fourier_bsk, uint32_t number_of_bits, uint32_t delta_log,
    uint32_t lwe_dimension_in, uint32_t lwe_dimension_out,
    uint32_t glwe_dimension, uint32_t base_log_bsk, uint32_t level_count_bsk,
    uint32_t base_log_ksk, uint32_t level_count_ksk, uint32_t number_of_samples,
    uint32_t max_shared_memory) {
  assert(("Error (GPU extract bits): base log should be <= 32",
          base_log_bsk <= 32));
  assert(("Error (GPU extract bits): glwe_dimension should be equal to 1",
          glwe_dimension == 1));
  assert(("Error (GPU extract bits): lwe_dimension_in should be one of "
          "512, 1024, 2048, 4096, 8192",
          lwe_dimension_in == 512 || lwe_dimension_in == 1024 ||
              lwe_dimension_in == 2048 || lwe_dimension_in == 4096 ||
              lwe_dimension_in == 8192));
  // The number of samples should be lower than 4 time the number of streaming
  // multiprocessors divided by ((k + 1) * l) (the factor 4 being related
  // to the occupancy of 50%). The only supported value for k is 1, so
  // k + 1 = 2 for now.
  int number_of_sm = 0;
  hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, 0);
  assert(("Error (GPU extract bits): the number of input LWEs must be lower or "
          "equal to the "
          "number of streaming multiprocessors on the device divided by 8 * "
          "level_count_bsk",
          number_of_samples <= number_of_sm * 4. / 2. / level_count_bsk));

  switch (lwe_dimension_in) {
  case 512:
    host_extract_bits<uint32_t, Degree<512>>(
        v_stream, gpu_index, (uint32_t *)list_lwe_array_out,
        (uint32_t *)lwe_array_in, (uint32_t *)lwe_array_in_buffer,
        (uint32_t *)lwe_array_in_shifted_buffer,
        (uint32_t *)lwe_array_out_ks_buffer,
        (uint32_t *)lwe_array_out_pbs_buffer, (uint32_t *)lut_pbs,
        (uint32_t *)lut_vector_indexes, (uint32_t *)ksk, (double2 *)fourier_bsk,
        number_of_bits, delta_log, lwe_dimension_in, lwe_dimension_out,
        base_log_bsk, level_count_bsk, base_log_ksk, level_count_ksk,
        number_of_samples, max_shared_memory);
    break;
  case 1024:
    host_extract_bits<uint32_t, Degree<1024>>(
        v_stream, gpu_index, (uint32_t *)list_lwe_array_out,
        (uint32_t *)lwe_array_in, (uint32_t *)lwe_array_in_buffer,
        (uint32_t *)lwe_array_in_shifted_buffer,
        (uint32_t *)lwe_array_out_ks_buffer,
        (uint32_t *)lwe_array_out_pbs_buffer, (uint32_t *)lut_pbs,
        (uint32_t *)lut_vector_indexes, (uint32_t *)ksk, (double2 *)fourier_bsk,
        number_of_bits, delta_log, lwe_dimension_in, lwe_dimension_out,
        base_log_bsk, level_count_bsk, base_log_ksk, level_count_ksk,
        number_of_samples, max_shared_memory);
    break;
  case 2048:
    host_extract_bits<uint32_t, Degree<2048>>(
        v_stream, gpu_index, (uint32_t *)list_lwe_array_out,
        (uint32_t *)lwe_array_in, (uint32_t *)lwe_array_in_buffer,
        (uint32_t *)lwe_array_in_shifted_buffer,
        (uint32_t *)lwe_array_out_ks_buffer,
        (uint32_t *)lwe_array_out_pbs_buffer, (uint32_t *)lut_pbs,
        (uint32_t *)lut_vector_indexes, (uint32_t *)ksk, (double2 *)fourier_bsk,
        number_of_bits, delta_log, lwe_dimension_in, lwe_dimension_out,
        base_log_bsk, level_count_bsk, base_log_ksk, level_count_ksk,
        number_of_samples, max_shared_memory);
    break;
  case 4096:
    host_extract_bits<uint32_t, Degree<4096>>(
        v_stream, gpu_index, (uint32_t *)list_lwe_array_out,
        (uint32_t *)lwe_array_in, (uint32_t *)lwe_array_in_buffer,
        (uint32_t *)lwe_array_in_shifted_buffer,
        (uint32_t *)lwe_array_out_ks_buffer,
        (uint32_t *)lwe_array_out_pbs_buffer, (uint32_t *)lut_pbs,
        (uint32_t *)lut_vector_indexes, (uint32_t *)ksk, (double2 *)fourier_bsk,
        number_of_bits, delta_log, lwe_dimension_in, lwe_dimension_out,
        base_log_bsk, level_count_bsk, base_log_ksk, level_count_ksk,
        number_of_samples, max_shared_memory);
    break;
  case 8192:
    host_extract_bits<uint32_t, Degree<8192>>(
        v_stream, gpu_index, (uint32_t *)list_lwe_array_out,
        (uint32_t *)lwe_array_in, (uint32_t *)lwe_array_in_buffer,
        (uint32_t *)lwe_array_in_shifted_buffer,
        (uint32_t *)lwe_array_out_ks_buffer,
        (uint32_t *)lwe_array_out_pbs_buffer, (uint32_t *)lut_pbs,
        (uint32_t *)lut_vector_indexes, (uint32_t *)ksk, (double2 *)fourier_bsk,
        number_of_bits, delta_log, lwe_dimension_in, lwe_dimension_out,
        base_log_bsk, level_count_bsk, base_log_ksk, level_count_ksk,
        number_of_samples, max_shared_memory);
    break;
  default:
    break;
  }
}

void cuda_extract_bits_64(
    void *v_stream, uint32_t gpu_index, void *list_lwe_array_out,
    void *lwe_array_in, void *lwe_array_in_buffer,
    void *lwe_array_in_shifted_buffer, void *lwe_array_out_ks_buffer,
    void *lwe_array_out_pbs_buffer, void *lut_pbs, void *lut_vector_indexes,
    void *ksk, void *fourier_bsk, uint32_t number_of_bits, uint32_t delta_log,
    uint32_t lwe_dimension_in, uint32_t lwe_dimension_out,
    uint32_t glwe_dimension, uint32_t base_log_bsk, uint32_t level_count_bsk,
    uint32_t base_log_ksk, uint32_t level_count_ksk, uint32_t number_of_samples,
    uint32_t max_shared_memory) {
  assert(("Error (GPU extract bits): base log should be <= 64",
          base_log_bsk <= 64));
  assert(("Error (GPU extract bits): glwe_dimension should be equal to 1",
          glwe_dimension == 1));
  assert(("Error (GPU extract bits): lwe_dimension_in should be one of "
          "512, 1024, 2048, 4096, 8192",
          lwe_dimension_in == 512 || lwe_dimension_in == 1024 ||
              lwe_dimension_in == 2048 || lwe_dimension_in == 4096 ||
              lwe_dimension_in == 8192));
  // The number of samples should be lower than four time the number of
  // streaming multiprocessors divided by (4 * (k + 1) * l) (the factor 4 being
  // related to the occupancy of 50%). The only supported value for k is 1, so
  // k + 1 = 2 for now.
  int number_of_sm = 0;
  hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, 0);
  assert(("Error (GPU extract bits): the number of input LWEs must be lower or "
          "equal to the "
          "number of streaming multiprocessors on the device divided by 8 * "
          "level_count_bsk",
          number_of_samples <= number_of_sm * 4. / 2. / level_count_bsk));

  switch (lwe_dimension_in) {
  case 512:
    host_extract_bits<uint64_t, Degree<512>>(
        v_stream, gpu_index, (uint64_t *)list_lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lwe_array_in_buffer,
        (uint64_t *)lwe_array_in_shifted_buffer,
        (uint64_t *)lwe_array_out_ks_buffer,
        (uint64_t *)lwe_array_out_pbs_buffer, (uint64_t *)lut_pbs,
        (uint32_t *)lut_vector_indexes, (uint64_t *)ksk, (double2 *)fourier_bsk,
        number_of_bits, delta_log, lwe_dimension_in, lwe_dimension_out,
        base_log_bsk, level_count_bsk, base_log_ksk, level_count_ksk,
        number_of_samples, max_shared_memory);
    break;
  case 1024:
    host_extract_bits<uint64_t, Degree<1024>>(
        v_stream, gpu_index, (uint64_t *)list_lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lwe_array_in_buffer,
        (uint64_t *)lwe_array_in_shifted_buffer,
        (uint64_t *)lwe_array_out_ks_buffer,
        (uint64_t *)lwe_array_out_pbs_buffer, (uint64_t *)lut_pbs,
        (uint32_t *)lut_vector_indexes, (uint64_t *)ksk, (double2 *)fourier_bsk,
        number_of_bits, delta_log, lwe_dimension_in, lwe_dimension_out,
        base_log_bsk, level_count_bsk, base_log_ksk, level_count_ksk,
        number_of_samples, max_shared_memory);
    break;
  case 2048:
    host_extract_bits<uint64_t, Degree<2048>>(
        v_stream, gpu_index, (uint64_t *)list_lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lwe_array_in_buffer,
        (uint64_t *)lwe_array_in_shifted_buffer,
        (uint64_t *)lwe_array_out_ks_buffer,
        (uint64_t *)lwe_array_out_pbs_buffer, (uint64_t *)lut_pbs,
        (uint32_t *)lut_vector_indexes, (uint64_t *)ksk, (double2 *)fourier_bsk,
        number_of_bits, delta_log, lwe_dimension_in, lwe_dimension_out,
        base_log_bsk, level_count_bsk, base_log_ksk, level_count_ksk,
        number_of_samples, max_shared_memory);
    break;
  case 4096:
    host_extract_bits<uint64_t, Degree<4096>>(
        v_stream, gpu_index, (uint64_t *)list_lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lwe_array_in_buffer,
        (uint64_t *)lwe_array_in_shifted_buffer,
        (uint64_t *)lwe_array_out_ks_buffer,
        (uint64_t *)lwe_array_out_pbs_buffer, (uint64_t *)lut_pbs,
        (uint32_t *)lut_vector_indexes, (uint64_t *)ksk, (double2 *)fourier_bsk,
        number_of_bits, delta_log, lwe_dimension_in, lwe_dimension_out,
        base_log_bsk, level_count_bsk, base_log_ksk, level_count_ksk,
        number_of_samples, max_shared_memory);
    break;
  case 8192:
    host_extract_bits<uint64_t, Degree<8192>>(
        v_stream, gpu_index, (uint64_t *)list_lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lwe_array_in_buffer,
        (uint64_t *)lwe_array_in_shifted_buffer,
        (uint64_t *)lwe_array_out_ks_buffer,
        (uint64_t *)lwe_array_out_pbs_buffer, (uint64_t *)lut_pbs,
        (uint32_t *)lut_vector_indexes, (uint64_t *)ksk, (double2 *)fourier_bsk,
        number_of_bits, delta_log, lwe_dimension_in, lwe_dimension_out,
        base_log_bsk, level_count_bsk, base_log_ksk, level_count_ksk,
        number_of_samples, max_shared_memory);
    break;
  default:
    break;
  }
}
