#include "hip/hip_runtime.h"
#include "bit_extraction.cuh"

/*
 * This scratch function allocates the necessary amount of data on the GPU for
 * the bit extraction on 32 bits inputs, into `cbs_buffer`. It also
 * configures SM options on the GPU in case FULLSM mode is going to be used.
 */
void scratch_cuda_extract_bits_32(
    void *v_stream, uint32_t gpu_index, int8_t **bit_extract_buffer,
    uint32_t glwe_dimension, uint32_t lwe_dimension, uint32_t polynomial_size,
    uint32_t level_count, uint32_t number_of_inputs, uint32_t max_shared_memory,
    bool allocate_gpu_memory) {

  switch (polynomial_size) {
  case 256:
    scratch_extract_bits<uint32_t, int32_t, Degree<256>>(
        v_stream, gpu_index, bit_extract_buffer, glwe_dimension, lwe_dimension,
        polynomial_size, level_count, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  case 512:
    scratch_extract_bits<uint32_t, int32_t, Degree<512>>(
        v_stream, gpu_index, bit_extract_buffer, glwe_dimension, lwe_dimension,
        polynomial_size, level_count, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  case 1024:
    scratch_extract_bits<uint32_t, int32_t, Degree<1024>>(
        v_stream, gpu_index, bit_extract_buffer, glwe_dimension, lwe_dimension,
        polynomial_size, level_count, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  case 2048:
    scratch_extract_bits<uint32_t, int32_t, Degree<2048>>(
        v_stream, gpu_index, bit_extract_buffer, glwe_dimension, lwe_dimension,
        polynomial_size, level_count, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  case 4096:
    scratch_extract_bits<uint32_t, int32_t, Degree<4096>>(
        v_stream, gpu_index, bit_extract_buffer, glwe_dimension, lwe_dimension,
        polynomial_size, level_count, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  case 8192:
    scratch_extract_bits<uint32_t, int32_t, Degree<8192>>(
        v_stream, gpu_index, bit_extract_buffer, glwe_dimension, lwe_dimension,
        polynomial_size, level_count, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  default:
    break;
  }
}

/*
 * This scratch function allocates the necessary amount of data on the GPU for
 * the bit extraction on 64 bits inputs, into `cbs_buffer`. It also
 * configures SM options on the GPU in case FULLSM mode is going to be used.
 */
void scratch_cuda_extract_bits_64(
    void *v_stream, uint32_t gpu_index, int8_t **bit_extract_buffer,
    uint32_t glwe_dimension, uint32_t lwe_dimension, uint32_t polynomial_size,
    uint32_t level_count, uint32_t number_of_inputs, uint32_t max_shared_memory,
    bool allocate_gpu_memory) {

  switch (polynomial_size) {
  case 256:
    scratch_extract_bits<uint64_t, int64_t, Degree<256>>(
        v_stream, gpu_index, bit_extract_buffer, glwe_dimension, lwe_dimension,
        polynomial_size, level_count, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  case 512:
    scratch_extract_bits<uint64_t, int64_t, Degree<512>>(
        v_stream, gpu_index, bit_extract_buffer, glwe_dimension, lwe_dimension,
        polynomial_size, level_count, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  case 1024:
    scratch_extract_bits<uint64_t, int64_t, Degree<1024>>(
        v_stream, gpu_index, bit_extract_buffer, glwe_dimension, lwe_dimension,
        polynomial_size, level_count, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  case 2048:
    scratch_extract_bits<uint64_t, int64_t, Degree<2048>>(
        v_stream, gpu_index, bit_extract_buffer, glwe_dimension, lwe_dimension,
        polynomial_size, level_count, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  case 4096:
    scratch_extract_bits<uint64_t, int64_t, Degree<4096>>(
        v_stream, gpu_index, bit_extract_buffer, glwe_dimension, lwe_dimension,
        polynomial_size, level_count, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  case 8192:
    scratch_extract_bits<uint64_t, int64_t, Degree<8192>>(
        v_stream, gpu_index, bit_extract_buffer, glwe_dimension, lwe_dimension,
        polynomial_size, level_count, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  default:
    break;
  }
}

/* Perform bit extract on a batch of 32 bit LWE ciphertexts.
 * See the corresponding function on 64 bit LWE ciphertexts for more details.
 */
void cuda_extract_bits_32(void *v_stream, uint32_t gpu_index,
                          void *list_lwe_array_out, void *lwe_array_in,
                          int8_t *bit_extract_buffer, void *ksk,
                          void *fourier_bsk, uint32_t number_of_bits,
                          uint32_t delta_log, uint32_t lwe_dimension_in,
                          uint32_t lwe_dimension_out, uint32_t glwe_dimension,
                          uint32_t polynomial_size, uint32_t base_log_bsk,
                          uint32_t level_count_bsk, uint32_t base_log_ksk,
                          uint32_t level_count_ksk, uint32_t number_of_samples,
                          uint32_t max_shared_memory) {
  assert(("Error (GPU extract bits): base log should be <= 32",
          base_log_bsk <= 32));
  assert(("Error (GPU extract bits): polynomial_size should be one of "
          "256, 512, 1024, 2048, 4096, 8192",
          polynomial_size == 256 || polynomial_size == 512 ||
              polynomial_size == 1024 || polynomial_size == 2048 ||
              polynomial_size == 4096 || polynomial_size == 8192));
  // The number of samples should be lower than four time the number of
  // streaming multiprocessors divided by (4 * (k + 1) * l) (the factor 4 being
  // related to the occupancy of 50%). The only supported value for k is 1, so
  // k + 1 = 2 for now.
  int number_of_sm = 0;
  hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, 0);
  assert(("Error (GPU extract bits): the number of input LWEs must be lower or "
          "equal to the "
          "number of streaming multiprocessors on the device divided by 8 * "
          "level_count_bsk",
          number_of_samples <= number_of_sm / 4. / 2. / level_count_bsk));

  switch (polynomial_size) {
  case 256:
    host_extract_bits<uint32_t, Degree<256>>(
        v_stream, gpu_index, (uint32_t *)list_lwe_array_out,
        (uint32_t *)lwe_array_in, bit_extract_buffer, (uint32_t *)ksk,
        (double2 *)fourier_bsk, number_of_bits, delta_log, lwe_dimension_in,
        lwe_dimension_out, glwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, number_of_samples,
        max_shared_memory);
    break;
  case 512:
    host_extract_bits<uint32_t, Degree<512>>(
        v_stream, gpu_index, (uint32_t *)list_lwe_array_out,
        (uint32_t *)lwe_array_in, bit_extract_buffer, (uint32_t *)ksk,
        (double2 *)fourier_bsk, number_of_bits, delta_log, lwe_dimension_in,
        lwe_dimension_out, glwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, number_of_samples,
        max_shared_memory);
    break;
  case 1024:
    host_extract_bits<uint32_t, Degree<1024>>(
        v_stream, gpu_index, (uint32_t *)list_lwe_array_out,
        (uint32_t *)lwe_array_in, bit_extract_buffer, (uint32_t *)ksk,
        (double2 *)fourier_bsk, number_of_bits, delta_log, lwe_dimension_in,
        lwe_dimension_out, glwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, number_of_samples,
        max_shared_memory);
    break;
  case 2048:
    host_extract_bits<uint32_t, Degree<2048>>(
        v_stream, gpu_index, (uint32_t *)list_lwe_array_out,
        (uint32_t *)lwe_array_in, bit_extract_buffer, (uint32_t *)ksk,
        (double2 *)fourier_bsk, number_of_bits, delta_log, lwe_dimension_in,
        lwe_dimension_out, glwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, number_of_samples,
        max_shared_memory);
    break;
  case 4096:
    host_extract_bits<uint32_t, Degree<4096>>(
        v_stream, gpu_index, (uint32_t *)list_lwe_array_out,
        (uint32_t *)lwe_array_in, bit_extract_buffer, (uint32_t *)ksk,
        (double2 *)fourier_bsk, number_of_bits, delta_log, lwe_dimension_in,
        lwe_dimension_out, glwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, number_of_samples,
        max_shared_memory);
    break;
  case 8192:
    host_extract_bits<uint32_t, Degree<8192>>(
        v_stream, gpu_index, (uint32_t *)list_lwe_array_out,
        (uint32_t *)lwe_array_in, bit_extract_buffer, (uint32_t *)ksk,
        (double2 *)fourier_bsk, number_of_bits, delta_log, lwe_dimension_in,
        lwe_dimension_out, glwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, number_of_samples,
        max_shared_memory);
    break;
  default:
    break;
  }
}

/* Perform bit extract on a batch of 64 bit lwe ciphertexts.
 * - `v_stream` is a void pointer to the Cuda stream to be used in the kernel
 * launch
 * - `gpu_index` is the index of the GPU to be used in the kernel launch
 *  - 'number_of_bits' will be extracted from each ciphertext
 * starting at the bit number 'delta_log' (0-indexed) included.
 * Output bits are ordered from the MSB to LSB. Every extracted bit is
 * represented as an LWE ciphertext, containing the encryption of the bit scaled
 * by q/2.
 *  - 'list_lwe_array_out' output batch LWE ciphertexts for each bit of every
 * input ciphertext
 *  - 'lwe_array_in' batch of input LWE ciphertexts, with size -
 * ('lwe_dimension_in' + 1) * number_of_samples * sizeof(u64)
 * The following 5 parameters are used during calculations, they are not actual
 * inputs of the function they are just allocated memory for calculation
 * process, like this, memory can be allocated once and can be used as much
 * as needed for different calls of extract_bit function.
 *  - 'lwe_array_in_buffer' same size as 'lwe_array_in'
 *  - 'lwe_array_in_shifted_buffer' same size as 'lwe_array_in'
 *  - 'lwe_array_out_ks_buffer'  with size:
 * ('lwe_dimension_out' + 1) * number_of_samples * sizeof(u64)
 *  - 'lwe_array_out_pbs_buffer' same size as 'lwe_array_in'
 *  - 'lut_pbs' with size:
 * (glwe_dimension + 1) * (lwe_dimension_in + 1) * sizeof(u64)
 * The other inputs are:
 *  - 'lut_vector_indexes' stores the index corresponding to which test
 * vector to use
 *  - 'ksk' keyswitch key
 *  - 'fourier_bsk'  complex compressed bsk in fourier domain
 *  - 'lwe_dimension_in' input LWE ciphertext dimension, supported input
 * dimensions are: {256, 512, 1024,2048, 4096, 8192}
 *  - 'lwe_dimension_out' output LWE ciphertext dimension
 *  - 'glwe_dimension' GLWE dimension,  only glwe_dimension = 1 is supported
 * for now
 *  - 'base_log_bsk' base_log for bootstrapping
 *  - 'level_count_bsk' decomposition level count for bootstrapping
 *  - 'base_log_ksk' base_log for keyswitch
 *  - 'level_count_ksk' decomposition level for keyswitch
 *  - 'number_of_samples' number of input LWE ciphertexts
 *  - 'max_shared_memory' maximum amount of shared memory to be used inside
 * device functions
 *
 * This function will call corresponding template of wrapper host function which
 * will manage the calls of device functions.
 */
void cuda_extract_bits_64(void *v_stream, uint32_t gpu_index,
                          void *list_lwe_array_out, void *lwe_array_in,
                          int8_t *bit_extract_buffer, void *ksk,
                          void *fourier_bsk, uint32_t number_of_bits,
                          uint32_t delta_log, uint32_t lwe_dimension_in,
                          uint32_t lwe_dimension_out, uint32_t glwe_dimension,
                          uint32_t polynomial_size, uint32_t base_log_bsk,
                          uint32_t level_count_bsk, uint32_t base_log_ksk,
                          uint32_t level_count_ksk, uint32_t number_of_samples,
                          uint32_t max_shared_memory) {
  assert(("Error (GPU extract bits): base log should be <= 64",
          base_log_bsk <= 64));
  assert(("Error (GPU extract bits): polynomial_size should be one of "
          "256, 512, 1024, 2048, 4096, 8192",
          polynomial_size == 256 || polynomial_size == 512 ||
              polynomial_size == 1024 || polynomial_size == 2048 ||
              polynomial_size == 4096 || polynomial_size == 8192));
  // The number of samples should be lower than four time the number of
  // streaming multiprocessors divided by (4 * (k + 1) * l) (the factor 4 being
  // related to the occupancy of 50%). The only supported value for k is 1, so
  // k + 1 = 2 for now.
  int number_of_sm = 0;
  hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, 0);
  assert(("Error (GPU extract bits): the number of input LWEs must be lower or "
          "equal to the "
          "number of streaming multiprocessors on the device divided by 8 * "
          "level_count_bsk",
          number_of_samples <= number_of_sm / 4. / 2. / level_count_bsk));

  switch (polynomial_size) {
  case 256:
    host_extract_bits<uint64_t, Degree<256>>(
        v_stream, gpu_index, (uint64_t *)list_lwe_array_out,
        (uint64_t *)lwe_array_in, bit_extract_buffer, (uint64_t *)ksk,
        (double2 *)fourier_bsk, number_of_bits, delta_log, lwe_dimension_in,
        lwe_dimension_out, glwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, number_of_samples,
        max_shared_memory);
    break;
  case 512:
    host_extract_bits<uint64_t, Degree<512>>(
        v_stream, gpu_index, (uint64_t *)list_lwe_array_out,
        (uint64_t *)lwe_array_in, bit_extract_buffer, (uint64_t *)ksk,
        (double2 *)fourier_bsk, number_of_bits, delta_log, lwe_dimension_in,
        lwe_dimension_out, glwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, number_of_samples,
        max_shared_memory);
    break;
  case 1024:
    host_extract_bits<uint64_t, Degree<1024>>(
        v_stream, gpu_index, (uint64_t *)list_lwe_array_out,
        (uint64_t *)lwe_array_in, bit_extract_buffer, (uint64_t *)ksk,
        (double2 *)fourier_bsk, number_of_bits, delta_log, lwe_dimension_in,
        lwe_dimension_out, glwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, number_of_samples,
        max_shared_memory);
    break;
  case 2048:
    host_extract_bits<uint64_t, Degree<2048>>(
        v_stream, gpu_index, (uint64_t *)list_lwe_array_out,
        (uint64_t *)lwe_array_in, bit_extract_buffer, (uint64_t *)ksk,
        (double2 *)fourier_bsk, number_of_bits, delta_log, lwe_dimension_in,
        lwe_dimension_out, glwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, number_of_samples,
        max_shared_memory);
    break;
  case 4096:
    host_extract_bits<uint64_t, Degree<4096>>(
        v_stream, gpu_index, (uint64_t *)list_lwe_array_out,
        (uint64_t *)lwe_array_in, bit_extract_buffer, (uint64_t *)ksk,
        (double2 *)fourier_bsk, number_of_bits, delta_log, lwe_dimension_in,
        lwe_dimension_out, glwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, number_of_samples,
        max_shared_memory);
    break;
  case 8192:
    host_extract_bits<uint64_t, Degree<8192>>(
        v_stream, gpu_index, (uint64_t *)list_lwe_array_out,
        (uint64_t *)lwe_array_in, bit_extract_buffer, (uint64_t *)ksk,
        (double2 *)fourier_bsk, number_of_bits, delta_log, lwe_dimension_in,
        lwe_dimension_out, glwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, number_of_samples,
        max_shared_memory);
    break;
  default:
    break;
  }
}

/*
 * This cleanup function frees the data for the bit extraction on GPU in
 * bit_extract_buffer for 32 or 64 bits inputs.
 */
void cleanup_cuda_extract_bits(void *v_stream, uint32_t gpu_index,
                               int8_t **bit_extract_buffer) {
  auto stream = static_cast<hipStream_t *>(v_stream);
  // Free memory
  cuda_drop_async(*bit_extract_buffer, stream, gpu_index);
}
