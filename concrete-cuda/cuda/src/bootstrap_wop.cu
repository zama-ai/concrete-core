#include "hip/hip_runtime.h"
#include "bootstrap_wop.cuh"

void cuda_cmux_tree_32(void *v_stream, void *glwe_out, void *ggsw_in,
                       void *lut_vector, uint32_t glwe_dimension,
                       uint32_t polynomial_size, uint32_t base_log,
                       uint32_t l_gadget, uint32_t r,
                       uint32_t max_shared_memory) {

  assert(("Error (GPU Cmux tree): base log should be <= 16", base_log <= 16));
  assert(("Error (GPU Cmux tree): polynomial size should be one of 512, 1024, "
          "2048, 4096, 8192",
          polynomial_size == 512 || polynomial_size == 1024 ||
              polynomial_size == 2048 || polynomial_size == 4096 ||
              polynomial_size == 8192));
  // For larger k we will need to adjust the mask size
  assert(("Error (GPU Cmux tree): glwe_dimension should be equal to 1",
          glwe_dimension == 1));
  assert(("Error (GPU Cmux tree): r, the number of layers in the tree, should "
          "be >= 1 ",
          r >= 1));

  switch (polynomial_size) {
  case 512:
    host_cmux_tree<uint32_t, int32_t, Degree<512>>(
        v_stream, (uint32_t *)glwe_out, (uint32_t *)ggsw_in,
        (uint32_t *)lut_vector, glwe_dimension, polynomial_size, base_log,
        l_gadget, r, max_shared_memory);
    break;
  case 1024:
    host_cmux_tree<uint32_t, int32_t, Degree<1024>>(
        v_stream, (uint32_t *)glwe_out, (uint32_t *)ggsw_in,
        (uint32_t *)lut_vector, glwe_dimension, polynomial_size, base_log,
        l_gadget, r, max_shared_memory);
    break;
  case 2048:
    host_cmux_tree<uint32_t, int32_t, Degree<2048>>(
        v_stream, (uint32_t *)glwe_out, (uint32_t *)ggsw_in,
        (uint32_t *)lut_vector, glwe_dimension, polynomial_size, base_log,
        l_gadget, r, max_shared_memory);
    break;
  case 4096:
    host_cmux_tree<uint32_t, int32_t, Degree<4096>>(
        v_stream, (uint32_t *)glwe_out, (uint32_t *)ggsw_in,
        (uint32_t *)lut_vector, glwe_dimension, polynomial_size, base_log,
        l_gadget, r, max_shared_memory);
    break;
  case 8192:
    host_cmux_tree<uint32_t, int32_t, Degree<8192>>(
        v_stream, (uint32_t *)glwe_out, (uint32_t *)ggsw_in,
        (uint32_t *)lut_vector, glwe_dimension, polynomial_size, base_log,
        l_gadget, r, max_shared_memory);
    break;
  default:
    break;
  }
}

void cuda_cmux_tree_64(void *v_stream, void *glwe_out, void *ggsw_in,
                       void *lut_vector, uint32_t glwe_dimension,
                       uint32_t polynomial_size, uint32_t base_log,
                       uint32_t l_gadget, uint32_t r,
                       uint32_t max_shared_memory) {

  assert(("Error (GPU Cmux tree): base log should be <= 16", base_log <= 16));
  assert(("Error (GPU Cmux tree): polynomial size should be one of 512, 1024, "
          "2048, 4096, 8192",
          polynomial_size == 512 || polynomial_size == 1024 ||
              polynomial_size == 2048 || polynomial_size == 4096 ||
              polynomial_size == 8192));
  // For larger k we will need to adjust the mask size
  assert(("Error (GPU Cmux tree): glwe_dimension should be equal to 1",
          glwe_dimension == 1));
  assert(("Error (GPU Cmux tree): r, the number of layers in the tree, should "
          "be >= 1 ",
          r >= 1));

  switch (polynomial_size) {
  case 512:
    host_cmux_tree<uint64_t, int64_t, Degree<512>>(
        v_stream, (uint64_t *)glwe_out, (uint64_t *)ggsw_in,
        (uint64_t *)lut_vector, glwe_dimension, polynomial_size, base_log,
        l_gadget, r, max_shared_memory);
    break;
  case 1024:
    host_cmux_tree<uint64_t, int64_t, Degree<1024>>(
        v_stream, (uint64_t *)glwe_out, (uint64_t *)ggsw_in,
        (uint64_t *)lut_vector, glwe_dimension, polynomial_size, base_log,
        l_gadget, r, max_shared_memory);
    break;
  case 2048:
    host_cmux_tree<uint64_t, int64_t, Degree<2048>>(
        v_stream, (uint64_t *)glwe_out, (uint64_t *)ggsw_in,
        (uint64_t *)lut_vector, glwe_dimension, polynomial_size, base_log,
        l_gadget, r, max_shared_memory);
    break;
  case 4096:
    host_cmux_tree<uint64_t, int64_t, Degree<4096>>(
        v_stream, (uint64_t *)glwe_out, (uint64_t *)ggsw_in,
        (uint64_t *)lut_vector, glwe_dimension, polynomial_size, base_log,
        l_gadget, r, max_shared_memory);
    break;
  case 8192:
    host_cmux_tree<uint64_t, int64_t, Degree<8192>>(
        v_stream, (uint64_t *)glwe_out, (uint64_t *)ggsw_in,
        (uint64_t *)lut_vector, glwe_dimension, polynomial_size, base_log,
        l_gadget, r, max_shared_memory);
    break;
  default:
    break;
  }
}

void cuda_extract_bits_32(void *v_stream, void *list_lwe_out, void *lwe_in,
                          void *lwe_in_buffer, void *lwe_in_shifted_buffer,
                          void *lwe_out_ks_buffer, void *lwe_out_pbs_buffer,
                          void *lut_pbs, void *lut_vector_indexes, void *ksk,
                          void *fourier_bsk, uint32_t number_of_bits,
                          uint32_t delta_log, uint32_t lwe_dimension_before,
                          uint32_t lwe_dimension_after, uint32_t glwe_dimension,
                          uint32_t base_log_bsk, uint32_t l_gadget_bsk,
                          uint32_t base_log_ksk, uint32_t l_gadget_ksk,
                          uint32_t number_of_samples) {
  assert(("Error (GPU extract bits): base log should be <= 16",
          base_log_bsk <= 16));
  assert(("Error (GPU extract bits): glwe_dimension should be equal to 1",
          glwe_dimension == 1));
  assert(("Error (GPU extract bits): lwe_dimension_before should be one of "
          "512, 1024, 2048",
          lwe_dimension_before == 512 || lwe_dimension_before == 1024 ||
              lwe_dimension_before == 2048));
  // The number of samples should be lower than the number of streaming
  // multiprocessors divided by (4 * (k + 1) * l) (the factor 4 being related
  // to the occupancy of 50%). The only supported value for k is 1, so
  // k + 1 = 2 for now.
  int number_of_sm = 0;
  hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, 0);
  assert(("Error (GPU extract bits): the number of input LWEs must be lower or "
          "equal to the "
          "number of streaming multiprocessors on the device divided by 8 * "
          "l_gadget_bsk",
          number_of_samples <= number_of_sm / 4. / 2. / l_gadget_bsk));

  switch (lwe_dimension_before) {
  case 512:
    host_extract_bits<uint32_t, Degree<512>>(
        v_stream, (uint32_t *)list_lwe_out, (uint32_t *)lwe_in,
        (uint32_t *)lwe_in_buffer, (uint32_t *)lwe_in_shifted_buffer,
        (uint32_t *)lwe_out_ks_buffer, (uint32_t *)lwe_out_pbs_buffer,
        (uint32_t *)lut_pbs, (uint32_t *)lut_vector_indexes, (uint32_t *)ksk,
        (double2 *)fourier_bsk, number_of_bits, delta_log, lwe_dimension_before,
        lwe_dimension_after, base_log_bsk, l_gadget_bsk, base_log_ksk,
        l_gadget_ksk, number_of_samples);
    break;
  case 1024:
    host_extract_bits<uint32_t, Degree<1024>>(
        v_stream, (uint32_t *)list_lwe_out, (uint32_t *)lwe_in,
        (uint32_t *)lwe_in_buffer, (uint32_t *)lwe_in_shifted_buffer,
        (uint32_t *)lwe_out_ks_buffer, (uint32_t *)lwe_out_pbs_buffer,
        (uint32_t *)lut_pbs, (uint32_t *)lut_vector_indexes, (uint32_t *)ksk,
        (double2 *)fourier_bsk, number_of_bits, delta_log, lwe_dimension_before,
        lwe_dimension_after, base_log_bsk, l_gadget_bsk, base_log_ksk,
        l_gadget_ksk, number_of_samples);
    break;
  case 2048:
    host_extract_bits<uint32_t, Degree<2048>>(
        v_stream, (uint32_t *)list_lwe_out, (uint32_t *)lwe_in,
        (uint32_t *)lwe_in_buffer, (uint32_t *)lwe_in_shifted_buffer,
        (uint32_t *)lwe_out_ks_buffer, (uint32_t *)lwe_out_pbs_buffer,
        (uint32_t *)lut_pbs, (uint32_t *)lut_vector_indexes, (uint32_t *)ksk,
        (double2 *)fourier_bsk, number_of_bits, delta_log, lwe_dimension_before,
        lwe_dimension_after, base_log_bsk, l_gadget_bsk, base_log_ksk,
        l_gadget_ksk, number_of_samples);
    break;
  default:
    break;
  }
}

void cuda_extract_bits_64(void *v_stream, void *list_lwe_out, void *lwe_in,
                          void *lwe_in_buffer, void *lwe_in_shifted_buffer,
                          void *lwe_out_ks_buffer, void *lwe_out_pbs_buffer,
                          void *lut_pbs, void *lut_vector_indexes, void *ksk,
                          void *fourier_bsk, uint32_t number_of_bits,
                          uint32_t delta_log, uint32_t lwe_dimension_before,
                          uint32_t lwe_dimension_after, uint32_t glwe_dimension,
                          uint32_t base_log_bsk, uint32_t l_gadget_bsk,
                          uint32_t base_log_ksk, uint32_t l_gadget_ksk,
                          uint32_t number_of_samples) {
  assert(("Error (GPU extract bits): base log should be <= 16",
          base_log_bsk <= 16));
  assert(("Error (GPU extract bits): glwe_dimension should be equal to 1",
          glwe_dimension == 1));
  assert(("Error (GPU extract bits): lwe_dimension_before should be one of "
          "512, 1024, 2048",
          lwe_dimension_before == 512 || lwe_dimension_before == 1024 ||
              lwe_dimension_before == 2048));
  // The number of samples should be lower than the number of streaming
  // multiprocessors divided by (4 * (k + 1) * l) (the factor 4 being related
  // to the occupancy of 50%). The only supported value for k is 1, so
  // k + 1 = 2 for now.
  int number_of_sm = 0;
  hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, 0);
  assert(("Error (GPU extract bits): the number of input LWEs must be lower or "
          "equal to the "
          "number of streaming multiprocessors on the device divided by 8 * "
          "l_gadget_bsk",
          number_of_samples <= number_of_sm / 4. / 2. / l_gadget_bsk));

  switch (lwe_dimension_before) {
  case 512:
    host_extract_bits<uint64_t, Degree<512>>(
        v_stream, (uint64_t *)list_lwe_out, (uint64_t *)lwe_in,
        (uint64_t *)lwe_in_buffer, (uint64_t *)lwe_in_shifted_buffer,
        (uint64_t *)lwe_out_ks_buffer, (uint64_t *)lwe_out_pbs_buffer,
        (uint64_t *)lut_pbs, (uint32_t *)lut_vector_indexes, (uint64_t *)ksk,
        (double2 *)fourier_bsk, number_of_bits, delta_log, lwe_dimension_before,
        lwe_dimension_after, base_log_bsk, l_gadget_bsk, base_log_ksk,
        l_gadget_ksk, number_of_samples);
    break;
  case 1024:
    host_extract_bits<uint64_t, Degree<1024>>(
        v_stream, (uint64_t *)list_lwe_out, (uint64_t *)lwe_in,
        (uint64_t *)lwe_in_buffer, (uint64_t *)lwe_in_shifted_buffer,
        (uint64_t *)lwe_out_ks_buffer, (uint64_t *)lwe_out_pbs_buffer,
        (uint64_t *)lut_pbs, (uint32_t *)lut_vector_indexes, (uint64_t *)ksk,
        (double2 *)fourier_bsk, number_of_bits, delta_log, lwe_dimension_before,
        lwe_dimension_after, base_log_bsk, l_gadget_bsk, base_log_ksk,
        l_gadget_ksk, number_of_samples);
    break;
  case 2048:
    host_extract_bits<uint64_t, Degree<2048>>(
        v_stream, (uint64_t *)list_lwe_out, (uint64_t *)lwe_in,
        (uint64_t *)lwe_in_buffer, (uint64_t *)lwe_in_shifted_buffer,
        (uint64_t *)lwe_out_ks_buffer, (uint64_t *)lwe_out_pbs_buffer,
        (uint64_t *)lut_pbs, (uint32_t *)lut_vector_indexes, (uint64_t *)ksk,
        (double2 *)fourier_bsk, number_of_bits, delta_log, lwe_dimension_before,
        lwe_dimension_after, base_log_bsk, l_gadget_bsk, base_log_ksk,
        l_gadget_ksk, number_of_samples);
    break;
  default:
    break;
  }
}
