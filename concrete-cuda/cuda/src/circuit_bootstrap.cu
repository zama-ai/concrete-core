#include "hip/hip_runtime.h"
#include "circuit_bootstrap.cuh"

/*
 * Perform circuit bootstrapping for the batch of 32 bit LWE ciphertexts.
 * Head out to the equivalent operation on 64 bits for more details.
 */
void cuda_circuit_bootstrap_32(
    void *v_stream, uint32_t gpu_index, void *ggsw_out, void *lwe_array_in,
    void *fourier_bsk, void *fp_ksk_array, void *lwe_array_in_shifted_buffer,
    void *lut_vector, void *lut_vector_indexes, void *lwe_array_out_pbs_buffer,
    void *lwe_array_in_fp_ks_buffer, uint32_t delta_log,
    uint32_t polynomial_size, uint32_t glwe_dimension, uint32_t lwe_dimension,
    uint32_t level_bsk, uint32_t base_log_bsk, uint32_t level_pksk,
    uint32_t base_log_pksk, uint32_t level_cbs, uint32_t base_log_cbs,
    uint32_t number_of_samples, uint32_t max_shared_memory) {
  assert(("Error (GPU circuit bootstrap): glwe_dimension should be equal to 1",
          glwe_dimension == 1));
  assert(("Error (GPU circuit bootstrap): polynomial_size should be one of "
          "512, 1024, 2048, 4096, 8192",
          polynomial_size == 512 || polynomial_size == 1024 ||
              polynomial_size == 2048 || polynomial_size == 4096 ||
              polynomial_size == 8192));
  // The number of samples should be lower than the number of streaming
  // multiprocessors divided by (4 * (k + 1) * l) (the factor 4 being related
  // to the occupancy of 50%). The only supported value for k is 1, so
  // k + 1 = 2 for now.
  int number_of_sm = 0;
  hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, 0);
  assert(("Error (GPU extract bits): the number of input LWEs must be lower or "
          "equal to the "
          "number of streaming multiprocessors on the device divided by 8 * "
          "level_count_bsk",
          number_of_samples <= number_of_sm / 4. / 2. / level_bsk));
  switch (polynomial_size) {
  case 512:
    host_circuit_bootstrap<uint32_t, Degree<512>>(
        v_stream, gpu_index, (uint32_t *)ggsw_out, (uint32_t *)lwe_array_in,
        (double2 *)fourier_bsk, (uint32_t *)fp_ksk_array,
        (uint32_t *)lwe_array_in_shifted_buffer, (uint32_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint32_t *)lwe_array_out_pbs_buffer,
        (uint32_t *)lwe_array_in_fp_ks_buffer, delta_log, polynomial_size,
        glwe_dimension, lwe_dimension, level_bsk, base_log_bsk, level_pksk,
        base_log_pksk, level_cbs, base_log_cbs, number_of_samples,
        max_shared_memory);
    break;
  case 1024:
    host_circuit_bootstrap<uint32_t, Degree<1024>>(
        v_stream, gpu_index, (uint32_t *)ggsw_out, (uint32_t *)lwe_array_in,
        (double2 *)fourier_bsk, (uint32_t *)fp_ksk_array,
        (uint32_t *)lwe_array_in_shifted_buffer, (uint32_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint32_t *)lwe_array_out_pbs_buffer,
        (uint32_t *)lwe_array_in_fp_ks_buffer, delta_log, polynomial_size,
        glwe_dimension, lwe_dimension, level_bsk, base_log_bsk, level_pksk,
        base_log_pksk, level_cbs, base_log_cbs, number_of_samples,
        max_shared_memory);
    break;
  case 2048:
    host_circuit_bootstrap<uint32_t, Degree<2048>>(
        v_stream, gpu_index, (uint32_t *)ggsw_out, (uint32_t *)lwe_array_in,
        (double2 *)fourier_bsk, (uint32_t *)fp_ksk_array,
        (uint32_t *)lwe_array_in_shifted_buffer, (uint32_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint32_t *)lwe_array_out_pbs_buffer,
        (uint32_t *)lwe_array_in_fp_ks_buffer, delta_log, polynomial_size,
        glwe_dimension, lwe_dimension, level_bsk, base_log_bsk, level_pksk,
        base_log_pksk, level_cbs, base_log_cbs, number_of_samples,
        max_shared_memory);
    break;
  case 4096:
    host_circuit_bootstrap<uint32_t, Degree<4096>>(
        v_stream, gpu_index, (uint32_t *)ggsw_out, (uint32_t *)lwe_array_in,
        (double2 *)fourier_bsk, (uint32_t *)fp_ksk_array,
        (uint32_t *)lwe_array_in_shifted_buffer, (uint32_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint32_t *)lwe_array_out_pbs_buffer,
        (uint32_t *)lwe_array_in_fp_ks_buffer, delta_log, polynomial_size,
        glwe_dimension, lwe_dimension, level_bsk, base_log_bsk, level_pksk,
        base_log_pksk, level_cbs, base_log_cbs, number_of_samples,
        max_shared_memory);
    break;
  case 8192:
    host_circuit_bootstrap<uint32_t, Degree<8192>>(
        v_stream, gpu_index, (uint32_t *)ggsw_out, (uint32_t *)lwe_array_in,
        (double2 *)fourier_bsk, (uint32_t *)fp_ksk_array,
        (uint32_t *)lwe_array_in_shifted_buffer, (uint32_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint32_t *)lwe_array_out_pbs_buffer,
        (uint32_t *)lwe_array_in_fp_ks_buffer, delta_log, polynomial_size,
        glwe_dimension, lwe_dimension, level_bsk, base_log_bsk, level_pksk,
        base_log_pksk, level_cbs, base_log_cbs, number_of_samples,
        max_shared_memory);
    break;
  default:
    break;
  }
}

/*
 * Perform circuit bootstrapping on a batch of 64 bit input LWE ciphertexts.
 * - `v_stream` is a void pointer to the Cuda stream to be used in the kernel
 * launch
 * - `gpu_index` is the index of the GPU to be used in the kernel launch
 *  - 'ggsw_out' output batch of ggsw with size:
 * 'number_of_samples' * 'level_cbs' * ('glwe_dimension' + 1)^2 *
 * polynomial_size * sizeof(u64)
 *  - 'lwe_array_in' input batch of lwe ciphertexts, with size:
 * 'number_of_samples' * '(lwe_dimension' + 1) * sizeof(u64)
 *  - 'fourier_bsk' bootstrapping key in fourier domain with size:
 * 'lwe_dimension' * 'level_bsk' * ('glwe_dimension' + 1)^2 *
 * 'polynomial_size' / 2 * sizeof(double2)
 *  - 'fp_ksk_array' batch of fp-keyswitch keys with size:
 * ('polynomial_size' + 1) * 'level_pksk' * ('glwe_dimension' + 1)^2 *
 * 'polynomial_size' * sizeof(u64)
 *  The following 5 parameters are used during calculations, they are not actual
 *  inputs of the function, they are just allocated memory for calculation
 *  process, like this, memory can be allocated once and can be used as much
 *  as needed for different calls of circuit_bootstrap function
 *  - 'lwe_array_in_shifted_buffer' with size:
 * 'number_of_samples' * 'level_cbs' * ('lwe_dimension' + 1) * sizeof(u64)
 *  - 'lut_vector' with size:
 * 'level_cbs' * ('glwe_dimension' + 1) * 'polynomial_size' * sizeof(u64)
 *  - 'lut_vector_indexes' stores the index corresponding to which test
 *  vector to use
 *  - 'lwe_array_out_pbs_buffer' with size
 * 'number_of_samples' * 'level_cbs' * ('polynomial_size' + 1) * sizeof(u64)
 *  - 'lwe_array_in_fp_ks_buffer' with size
 * 'number_of_samples' * 'level_cbs' * ('glwe_dimension' + 1) *
 * ('polynomial_size' + 1) * sizeof(u64)
 *
 * This function calls a wrapper to a device kernel that performs the
 * circuit bootstrap. The kernel is templatized based on integer discretization
 * and polynomial degree.
 */
void cuda_circuit_bootstrap_64(
    void *v_stream, uint32_t gpu_index, void *ggsw_out, void *lwe_array_in,
    void *fourier_bsk, void *fp_ksk_array, void *lwe_array_in_shifted_buffer,
    void *lut_vector, void *lut_vector_indexes, void *lwe_array_out_pbs_buffer,
    void *lwe_array_in_fp_ks_buffer, uint32_t delta_log,
    uint32_t polynomial_size, uint32_t glwe_dimension, uint32_t lwe_dimension,
    uint32_t level_bsk, uint32_t base_log_bsk, uint32_t level_pksk,
    uint32_t base_log_pksk, uint32_t level_cbs, uint32_t base_log_cbs,
    uint32_t number_of_samples, uint32_t max_shared_memory) {
  assert(("Error (GPU circuit bootstrap): glwe_dimension should be equal to 1",
          glwe_dimension == 1));
  assert(("Error (GPU circuit bootstrap): polynomial_size should be one of "
          "512, 1024, 2048, 4096, 8192",
          polynomial_size == 512 || polynomial_size == 1024 ||
              polynomial_size == 2048 || polynomial_size == 4096 ||
              polynomial_size == 8192));
  // The number of samples should be lower than the number of streaming
  // multiprocessors divided by (4 * (k + 1) * l) (the factor 4 being related
  // to the occupancy of 50%). The only supported value for k is 1, so
  // k + 1 = 2 for now.
  int number_of_sm = 0;
  hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, 0);
  assert(("Error (GPU extract bits): the number of input LWEs must be lower or "
          "equal to the "
          "number of streaming multiprocessors on the device divided by 8 * "
          "level_count_bsk",
          number_of_samples <= number_of_sm / 4. / 2. / level_bsk));
  // The number of samples should be lower than the number of streaming
  switch (polynomial_size) {
  case 512:
    host_circuit_bootstrap<uint64_t, Degree<512>>(
        v_stream, gpu_index, (uint64_t *)ggsw_out, (uint64_t *)lwe_array_in,
        (double2 *)fourier_bsk, (uint64_t *)fp_ksk_array,
        (uint64_t *)lwe_array_in_shifted_buffer, (uint64_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint64_t *)lwe_array_out_pbs_buffer,
        (uint64_t *)lwe_array_in_fp_ks_buffer, delta_log, polynomial_size,
        glwe_dimension, lwe_dimension, level_bsk, base_log_bsk, level_pksk,
        base_log_pksk, level_cbs, base_log_cbs, number_of_samples,
        max_shared_memory);
    break;
  case 1024:
    host_circuit_bootstrap<uint64_t, Degree<1024>>(
        v_stream, gpu_index, (uint64_t *)ggsw_out, (uint64_t *)lwe_array_in,
        (double2 *)fourier_bsk, (uint64_t *)fp_ksk_array,
        (uint64_t *)lwe_array_in_shifted_buffer, (uint64_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint64_t *)lwe_array_out_pbs_buffer,
        (uint64_t *)lwe_array_in_fp_ks_buffer, delta_log, polynomial_size,
        glwe_dimension, lwe_dimension, level_bsk, base_log_bsk, level_pksk,
        base_log_pksk, level_cbs, base_log_cbs, number_of_samples,
        max_shared_memory);
    break;
  case 2048:
    host_circuit_bootstrap<uint64_t, Degree<2048>>(
        v_stream, gpu_index, (uint64_t *)ggsw_out, (uint64_t *)lwe_array_in,
        (double2 *)fourier_bsk, (uint64_t *)fp_ksk_array,
        (uint64_t *)lwe_array_in_shifted_buffer, (uint64_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint64_t *)lwe_array_out_pbs_buffer,
        (uint64_t *)lwe_array_in_fp_ks_buffer, delta_log, polynomial_size,
        glwe_dimension, lwe_dimension, level_bsk, base_log_bsk, level_pksk,
        base_log_pksk, level_cbs, base_log_cbs, number_of_samples,
        max_shared_memory);
    break;
  case 4096:
    host_circuit_bootstrap<uint64_t, Degree<4096>>(
        v_stream, gpu_index, (uint64_t *)ggsw_out, (uint64_t *)lwe_array_in,
        (double2 *)fourier_bsk, (uint64_t *)fp_ksk_array,
        (uint64_t *)lwe_array_in_shifted_buffer, (uint64_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint64_t *)lwe_array_out_pbs_buffer,
        (uint64_t *)lwe_array_in_fp_ks_buffer, delta_log, polynomial_size,
        glwe_dimension, lwe_dimension, level_bsk, base_log_bsk, level_pksk,
        base_log_pksk, level_cbs, base_log_cbs, number_of_samples,
        max_shared_memory);
    break;
  case 8192:
    host_circuit_bootstrap<uint64_t, Degree<8192>>(
        v_stream, gpu_index, (uint64_t *)ggsw_out, (uint64_t *)lwe_array_in,
        (double2 *)fourier_bsk, (uint64_t *)fp_ksk_array,
        (uint64_t *)lwe_array_in_shifted_buffer, (uint64_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint64_t *)lwe_array_out_pbs_buffer,
        (uint64_t *)lwe_array_in_fp_ks_buffer, delta_log, polynomial_size,
        glwe_dimension, lwe_dimension, level_bsk, base_log_bsk, level_pksk,
        base_log_pksk, level_cbs, base_log_cbs, number_of_samples,
        max_shared_memory);
    break;
  default:
    break;
  }
}
