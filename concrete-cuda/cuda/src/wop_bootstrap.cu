#include "hip/hip_runtime.h"
#include "wop_bootstrap.cuh"

void cuda_circuit_bootstrap_vertical_packing_32(
    void *v_stream, uint32_t gpu_index, void *lwe_array_out, void *lwe_array_in,
    void *fourier_bsk, void *cbs_fpksk, void *lut_vector,
    uint32_t polynomial_size, uint32_t glwe_dimension, uint32_t lwe_dimension,
    uint32_t level_count_bsk, uint32_t base_log_bsk, uint32_t level_count_pksk,
    uint32_t base_log_pksk, uint32_t level_count_cbs, uint32_t base_log_cbs,
    uint32_t number_of_inputs, uint32_t lut_number,
    uint32_t max_shared_memory) {
  assert(("Error (GPU circuit bootstrap): glwe_dimension should be equal to 1",
          glwe_dimension == 1));
  assert(("Error (GPU circuit bootstrap): polynomial_size should be one of "
          "512, 1024, 2048, 4096, 8192",
          polynomial_size == 512 || polynomial_size == 1024 ||
              polynomial_size == 2048 || polynomial_size == 4096 ||
              polynomial_size == 8192));
  // The number of inputs should be lower than the number of streaming
  // multiprocessors divided by (4 * (k + 1) * l) (the factor 4 being related
  // to the occupancy of 50%). The only supported value for k is 1, so
  // k + 1 = 2 for now.
  int number_of_sm = 0;
  hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, 0);
  assert(("Error (GPU extract bits): the number of input LWEs must be lower or "
          "equal to the "
          "number of streaming multiprocessors on the device divided by 8 * "
          "level_count_bsk",
          number_of_inputs <= number_of_sm / 4. / 2. / level_count_bsk));
  switch (polynomial_size) {
  case 512:
    host_circuit_bootstrap_vertical_packing<uint32_t, int32_t, Degree<512>>(
        v_stream, gpu_index, (uint32_t *)lwe_array_out,
        (uint32_t *)lwe_array_in, (uint32_t *)lut_vector,
        (double2 *)fourier_bsk, (uint32_t *)cbs_fpksk, glwe_dimension,
        lwe_dimension, polynomial_size, base_log_bsk, level_count_bsk,
        base_log_pksk, level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_inputs, lut_number, max_shared_memory);
    break;
  case 1024:
    host_circuit_bootstrap_vertical_packing<uint32_t, int32_t, Degree<1024>>(
        v_stream, gpu_index, (uint32_t *)lwe_array_out,
        (uint32_t *)lwe_array_in, (uint32_t *)lut_vector,
        (double2 *)fourier_bsk, (uint32_t *)cbs_fpksk, glwe_dimension,
        lwe_dimension, polynomial_size, base_log_bsk, level_count_bsk,
        base_log_pksk, level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_inputs, lut_number, max_shared_memory);
    break;
  case 2048:
    host_circuit_bootstrap_vertical_packing<uint32_t, int32_t, Degree<2048>>(
        v_stream, gpu_index, (uint32_t *)lwe_array_out,
        (uint32_t *)lwe_array_in, (uint32_t *)lut_vector,
        (double2 *)fourier_bsk, (uint32_t *)cbs_fpksk, glwe_dimension,
        lwe_dimension, polynomial_size, base_log_bsk, level_count_bsk,
        base_log_pksk, level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_inputs, lut_number, max_shared_memory);
    break;
  case 4096:
    host_circuit_bootstrap_vertical_packing<uint32_t, int32_t, Degree<4096>>(
        v_stream, gpu_index, (uint32_t *)lwe_array_out,
        (uint32_t *)lwe_array_in, (uint32_t *)lut_vector,
        (double2 *)fourier_bsk, (uint32_t *)cbs_fpksk, glwe_dimension,
        lwe_dimension, polynomial_size, base_log_bsk, level_count_bsk,
        base_log_pksk, level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_inputs, lut_number, max_shared_memory);
    break;
  case 8192:
    host_circuit_bootstrap_vertical_packing<uint32_t, int32_t, Degree<8192>>(
        v_stream, gpu_index, (uint32_t *)lwe_array_out,
        (uint32_t *)lwe_array_in, (uint32_t *)lut_vector,
        (double2 *)fourier_bsk, (uint32_t *)cbs_fpksk, glwe_dimension,
        lwe_dimension, polynomial_size, base_log_bsk, level_count_bsk,
        base_log_pksk, level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_inputs, lut_number, max_shared_memory);
    break;
  default:
    break;
  }
}

/*
 * Entry point for cuda circuit bootstrap + vertical packing for batches of
 * input 64 bit LWE ciphertexts.
 *  - `v_stream` is a void pointer to the Cuda stream to be used in the kernel
 * launch
 *  - `gpu_index` is the index of the GPU to be used in the kernel launch
 *  - 'lwe_array_out' list of output lwe ciphertexts
 *  - 'lwe_array_in' list of input lwe_ciphertexts
 *  - 'fourier_bsk' bootstrapping key in fourier domain, expected half size
 * compressed complex key.
 *  - 'cbs_fpksk' list of private functional packing keyswitch keys
 *  - 'lut_vector' list of test vectors
 *  - 'polynomial_size' size of the test polynomial, supported sizes:
 * {512, 1024, 2048, 4096, 8192}
 *  - 'glwe_dimension' supported dimensions: {1}
 *  - 'lwe_dimension' dimension of input LWE ciphertexts
 *  - 'level_count_bsk' decomposition level for bootstrapping
 *  - 'base_log_bsk'  base log parameter for bootstrapping
 *  - 'level_count_pksk' decomposition level for fp-keyswitch
 *  - 'base_log_pksk' base log parameter for fp-keyswitch
 *  - 'level_count_cbs' level of circuit bootstrap
 *  - 'base_log_cbs' base log parameter for circuit bootstrap
 *  - 'number_of_inputs' number of input LWE ciphertexts
 *  - 'max_shared_memory' maximum shared memory amount to be used in
 *  bootstrapping kernel
 *
 */
void cuda_circuit_bootstrap_vertical_packing_64(
    void *v_stream, uint32_t gpu_index, void *lwe_array_out, void *lwe_array_in,
    void *fourier_bsk, void *cbs_fpksk, void *lut_vector,
    uint32_t polynomial_size, uint32_t glwe_dimension, uint32_t lwe_dimension,
    uint32_t level_count_bsk, uint32_t base_log_bsk, uint32_t level_count_pksk,
    uint32_t base_log_pksk, uint32_t level_count_cbs, uint32_t base_log_cbs,
    uint32_t number_of_inputs, uint32_t lut_number,
    uint32_t max_shared_memory) {
  assert(("Error (GPU circuit bootstrap): glwe_dimension should be equal to 1",
          glwe_dimension == 1));
  assert(("Error (GPU circuit bootstrap): polynomial_size should be one of "
          "512, 1024, 2048, 4096, 8192",
          polynomial_size == 512 || polynomial_size == 1024 ||
              polynomial_size == 2048 || polynomial_size == 4096 ||
              polynomial_size == 8192));
  // The number of inputs should be lower than the number of streaming
  // multiprocessors divided by (4 * (k + 1) * l) (the factor 4 being related
  // to the occupancy of 50%). The only supported value for k is 1, so
  // k + 1 = 2 for now.
  int number_of_sm = 0;
  hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, 0);
  assert(("Error (GPU extract bits): the number of input LWEs must be lower or "
          "equal to the "
          "number of streaming multiprocessors on the device divided by 8 * "
          "level_count_bsk",
          number_of_inputs <= number_of_sm / 4. / 2. / level_count_bsk));
  switch (polynomial_size) {
  case 512:
    host_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<512>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)cbs_fpksk, glwe_dimension,
        lwe_dimension, polynomial_size, base_log_bsk, level_count_bsk,
        base_log_pksk, level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_inputs, lut_number, max_shared_memory);
    break;
  case 1024:
    host_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<1024>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)cbs_fpksk, glwe_dimension,
        lwe_dimension, polynomial_size, base_log_bsk, level_count_bsk,
        base_log_pksk, level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_inputs, lut_number, max_shared_memory);
    break;
  case 2048:
    host_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<2048>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)cbs_fpksk, glwe_dimension,
        lwe_dimension, polynomial_size, base_log_bsk, level_count_bsk,
        base_log_pksk, level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_inputs, lut_number, max_shared_memory);
    break;
  case 4096:
    host_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<4096>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)cbs_fpksk, glwe_dimension,
        lwe_dimension, polynomial_size, base_log_bsk, level_count_bsk,
        base_log_pksk, level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_inputs, lut_number, max_shared_memory);
    break;
  case 8192:
    host_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<8192>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)cbs_fpksk, glwe_dimension,
        lwe_dimension, polynomial_size, base_log_bsk, level_count_bsk,
        base_log_pksk, level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_inputs, lut_number, max_shared_memory);
    break;
  default:
    break;
  }
}

/*
 * Entry point for entire without padding programmable bootstrap on 64 bit input
 * LWE ciphertexts.
 * See the equivalent operation on 64 bit inputs for more details.
 */
void cuda_wop_pbs_32(void *v_stream, uint32_t gpu_index, void *lwe_array_out,
                     void *lwe_array_in, void *lut_vector, void *fourier_bsk,
                     void *ksk, void *cbs_fpksk, uint32_t glwe_dimension,
                     uint32_t lwe_dimension, uint32_t polynomial_size,
                     uint32_t base_log_bsk, uint32_t level_count_bsk,
                     uint32_t base_log_ksk, uint32_t level_count_ksk,
                     uint32_t base_log_pksk, uint32_t level_count_pksk,
                     uint32_t base_log_cbs, uint32_t level_count_cbs,
                     uint32_t number_of_bits_of_message_including_padding,
                     uint32_t number_of_bits_to_extract,
                     uint32_t number_of_inputs, uint32_t max_shared_memory) {
  assert(("Error (GPU WOP PBS): glwe_dimension should be equal to 1",
          glwe_dimension == 1));
  assert(("Error (GPU WOP PBS): polynomial_size should be one of "
          "512, 1024, 2048, 4096, 8192",
          polynomial_size == 512 || polynomial_size == 1024 ||
              polynomial_size == 2048 || polynomial_size == 4096 ||
              polynomial_size == 8192));
  // The number of inputs should be lower than the number of streaming
  // multiprocessors divided by (4 * (k + 1) * l) (the factor 4 being related
  // to the occupancy of 50%). The only supported value for k is 1, so
  // k + 1 = 2 for now.
  int number_of_sm = 0;
  hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, 0);
  assert(("Error (GPU WOP PBS): the number of input LWEs must be lower or "
          "equal to the "
          "number of streaming multiprocessors on the device divided by 8 * "
          "level_count_bsk",
          number_of_inputs <= number_of_sm / 4. / 2. / level_count_bsk));
  switch (polynomial_size) {
  case 512:
    host_wop_pbs<uint32_t, int32_t, Degree<512>>(
        v_stream, gpu_index, (uint32_t *)lwe_array_out,
        (uint32_t *)lwe_array_in, (uint32_t *)lut_vector,
        (double2 *)fourier_bsk, (uint32_t *)ksk, (uint32_t *)cbs_fpksk,
        glwe_dimension, lwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, base_log_pksk,
        level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_bits_of_message_including_padding, number_of_bits_to_extract,
        number_of_inputs, max_shared_memory);
    break;
  case 1024:
    host_wop_pbs<uint32_t, int32_t, Degree<1024>>(
        v_stream, gpu_index, (uint32_t *)lwe_array_out,
        (uint32_t *)lwe_array_in, (uint32_t *)lut_vector,
        (double2 *)fourier_bsk, (uint32_t *)ksk, (uint32_t *)cbs_fpksk,
        glwe_dimension, lwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, base_log_pksk,
        level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_bits_of_message_including_padding, number_of_bits_to_extract,
        number_of_inputs, max_shared_memory);
    break;
  case 2048:
    host_wop_pbs<uint32_t, int32_t, Degree<2048>>(
        v_stream, gpu_index, (uint32_t *)lwe_array_out,
        (uint32_t *)lwe_array_in, (uint32_t *)lut_vector,
        (double2 *)fourier_bsk, (uint32_t *)ksk, (uint32_t *)cbs_fpksk,
        glwe_dimension, lwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, base_log_pksk,
        level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_bits_of_message_including_padding, number_of_bits_to_extract,
        number_of_inputs, max_shared_memory);
    break;
  case 4096:
    host_wop_pbs<uint32_t, int32_t, Degree<4096>>(
        v_stream, gpu_index, (uint32_t *)lwe_array_out,
        (uint32_t *)lwe_array_in, (uint32_t *)lut_vector,
        (double2 *)fourier_bsk, (uint32_t *)ksk, (uint32_t *)cbs_fpksk,
        glwe_dimension, lwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, base_log_pksk,
        level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_bits_of_message_including_padding, number_of_bits_to_extract,
        number_of_inputs, max_shared_memory);
    break;
  case 8192:
    host_wop_pbs<uint32_t, int32_t, Degree<8192>>(
        v_stream, gpu_index, (uint32_t *)lwe_array_out,
        (uint32_t *)lwe_array_in, (uint32_t *)lut_vector,
        (double2 *)fourier_bsk, (uint32_t *)ksk, (uint32_t *)cbs_fpksk,
        glwe_dimension, lwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, base_log_pksk,
        level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_bits_of_message_including_padding, number_of_bits_to_extract,
        number_of_inputs, max_shared_memory);
    break;
  default:
    break;
  }
}

/*
 * Entry point for entire without padding programmable bootstrap on 64 bit input
 * LWE ciphertexts.
 *  - `v_stream` is a void pointer to the Cuda stream to be used in the kernel
 * launch
 *  - `gpu_index` is the index of the GPU to be used in the kernel launch
 *  - 'lwe_array_out' list of output lwe ciphertexts
 *  - 'lwe_array_in' list of input lwe_ciphertexts
 *  - 'lut_vector' list of test vectors
 *  - 'fourier_bsk' bootstrapping key in fourier domain, expected half size
 * compressed complex key.
 *  - 'ksk' keyswitch key to use inside extract bits block
 *  - 'cbs_fpksk' list of fp-keyswitch keys
 *  - 'glwe_dimension' supported dimensions: {1}
 *  - 'lwe_dimension' dimension of input lwe ciphertexts
 *  - 'polynomial_size' size of the test polynomial, supported sizes:
 * {512, 1024, 2048, 4096, 8192}
 *  - 'base_log_bsk'  base log parameter for bootstrapping
 *  - 'level_count_bsk' decomposition level for bootstrapping
 *  - 'base_log_ksk' base log parameter for keyswitch
 *  - 'level_count_ksk' decomposition level for keyswitch
 *  - 'base_log_pksk' base log parameter for fp-keyswitch
 *  - 'level_count_pksk' decomposition level for fp-keyswitch
 *  - 'base_log_cbs' base log parameter for circuit bootstrap
 *  - 'level_count_cbs' level of circuit bootstrap
 *  - 'number_of_bits_of_message_including_padding' number of bits to extract
 * from each input lwe ciphertext including padding bit
 *  - 'number_of_bits_to_extract' number of bits to extract
 * from each input lwe ciphertext without padding bit
 *  - 'number_of_inputs' number of input lwe ciphertexts
 *  - 'max_shared_memory' maximum shared memory amount to be used in
 *  bootstrapping kernel
 *
 */
void cuda_wop_pbs_64(void *v_stream, uint32_t gpu_index, void *lwe_array_out,
                     void *lwe_array_in, void *lut_vector, void *fourier_bsk,
                     void *ksk, void *cbs_fpksk, uint32_t glwe_dimension,
                     uint32_t lwe_dimension, uint32_t polynomial_size,
                     uint32_t base_log_bsk, uint32_t level_count_bsk,
                     uint32_t base_log_ksk, uint32_t level_count_ksk,
                     uint32_t base_log_pksk, uint32_t level_count_pksk,
                     uint32_t base_log_cbs, uint32_t level_count_cbs,
                     uint32_t number_of_bits_of_message_including_padding,
                     uint32_t number_of_bits_to_extract,
                     uint32_t number_of_inputs, uint32_t max_shared_memory) {
  assert(("Error (GPU WOP PBS): glwe_dimension should be equal to 1",
          glwe_dimension == 1));
  assert(("Error (GPU WOP PBS): polynomial_size should be one of "
          "512, 1024, 2048, 4096, 8192",
          polynomial_size == 512 || polynomial_size == 1024 ||
              polynomial_size == 2048 || polynomial_size == 4096 ||
              polynomial_size == 8192));
  // The number of inputs should be lower than the number of streaming
  // multiprocessors divided by (4 * (k + 1) * l) (the factor 4 being related
  // to the occupancy of 50%). The only supported value for k is 1, so
  // k + 1 = 2 for now.
  int number_of_sm = 0;
  hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, 0);
  assert(("Error (GPU WOP PBS): the number of input LWEs must be lower or "
          "equal to the "
          "number of streaming multiprocessors on the device divided by 8 * "
          "level_count_bsk",
          number_of_inputs <= number_of_sm / 4. / 2. / level_count_bsk));
  switch (polynomial_size) {
  case 512:
    host_wop_pbs<uint64_t, int64_t, Degree<512>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)ksk, (uint64_t *)cbs_fpksk,
        glwe_dimension, lwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, base_log_pksk,
        level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_bits_of_message_including_padding, number_of_bits_to_extract,
        number_of_inputs, max_shared_memory);
    break;
  case 1024:
    host_wop_pbs<uint64_t, int64_t, Degree<1024>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)ksk, (uint64_t *)cbs_fpksk,
        glwe_dimension, lwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, base_log_pksk,
        level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_bits_of_message_including_padding, number_of_bits_to_extract,
        number_of_inputs, max_shared_memory);
    break;
  case 2048:
    host_wop_pbs<uint64_t, int64_t, Degree<2048>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)ksk, (uint64_t *)cbs_fpksk,
        glwe_dimension, lwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, base_log_pksk,
        level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_bits_of_message_including_padding, number_of_bits_to_extract,
        number_of_inputs, max_shared_memory);
    break;
  case 4096:
    host_wop_pbs<uint64_t, int64_t, Degree<4096>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)ksk, (uint64_t *)cbs_fpksk,
        glwe_dimension, lwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, base_log_pksk,
        level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_bits_of_message_including_padding, number_of_bits_to_extract,
        number_of_inputs, max_shared_memory);
    break;
  case 8192:
    host_wop_pbs<uint64_t, int64_t, Degree<8192>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)ksk, (uint64_t *)cbs_fpksk,
        glwe_dimension, lwe_dimension, polynomial_size, base_log_bsk,
        level_count_bsk, base_log_ksk, level_count_ksk, base_log_pksk,
        level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_bits_of_message_including_padding, number_of_bits_to_extract,
        number_of_inputs, max_shared_memory);
    break;
  default:
    break;
  }
}
