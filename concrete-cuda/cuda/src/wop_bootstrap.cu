#include "hip/hip_runtime.h"
#include "wop_bootstrap.cuh"

/*
 * This scratch function allocates the necessary amount of data on the GPU for
 * the circuit bootstrap and vertical packing on 32 bits inputs, into
 * `cbs_vp_buffer`. It also fills the value of delta_log to be used in the
 * circuit bootstrap.
 */
void scratch_cuda_circuit_bootstrap_vertical_packing_32(
    void *v_stream, uint32_t gpu_index, int8_t **cbs_vp_buffer,
    uint32_t *cbs_delta_log, uint32_t glwe_dimension, uint32_t lwe_dimension,
    uint32_t polynomial_size, uint32_t level_count_cbs,
    uint32_t number_of_inputs, uint32_t tau, uint32_t max_shared_memory,
    bool allocate_gpu_memory) {

  switch (polynomial_size) {
  case 512:
    scratch_circuit_bootstrap_vertical_packing<uint32_t, int32_t, Degree<512>>(
        v_stream, gpu_index, cbs_vp_buffer, cbs_delta_log, glwe_dimension,
        lwe_dimension, polynomial_size, level_count_cbs, number_of_inputs, tau,
        max_shared_memory, allocate_gpu_memory);
    break;
  case 1024:
    scratch_circuit_bootstrap_vertical_packing<uint32_t, int32_t, Degree<1024>>(
        v_stream, gpu_index, cbs_vp_buffer, cbs_delta_log, glwe_dimension,
        lwe_dimension, polynomial_size, level_count_cbs, number_of_inputs, tau,
        max_shared_memory, allocate_gpu_memory);
    break;
  case 2048:
    scratch_circuit_bootstrap_vertical_packing<uint32_t, int32_t, Degree<2048>>(
        v_stream, gpu_index, cbs_vp_buffer, cbs_delta_log, glwe_dimension,
        lwe_dimension, polynomial_size, level_count_cbs, number_of_inputs, tau,
        max_shared_memory, allocate_gpu_memory);
    break;
  case 4096:
    scratch_circuit_bootstrap_vertical_packing<uint32_t, int32_t, Degree<4096>>(
        v_stream, gpu_index, cbs_vp_buffer, cbs_delta_log, glwe_dimension,
        lwe_dimension, polynomial_size, level_count_cbs, number_of_inputs, tau,
        max_shared_memory, allocate_gpu_memory);
    break;
  case 8192:
    scratch_circuit_bootstrap_vertical_packing<uint32_t, int32_t, Degree<8192>>(
        v_stream, gpu_index, cbs_vp_buffer, cbs_delta_log, glwe_dimension,
        lwe_dimension, polynomial_size, level_count_cbs, number_of_inputs, tau,
        max_shared_memory, allocate_gpu_memory);
    break;
  default:
    break;
  }
}

/*
 * This scratch function allocates the necessary amount of data on the GPU for
 * the circuit bootstrap and vertical packing on 64 bits inputs, into
 * `cbs_vp_buffer`. It also fills the value of delta_log to be used in the
 * circuit bootstrap.
 */
void scratch_cuda_circuit_bootstrap_vertical_packing_64(
    void *v_stream, uint32_t gpu_index, int8_t **cbs_vp_buffer,
    uint32_t *cbs_delta_log, uint32_t glwe_dimension, uint32_t lwe_dimension,
    uint32_t polynomial_size, uint32_t level_count_cbs,
    uint32_t number_of_inputs, uint32_t tau, uint32_t max_shared_memory,
    bool allocate_gpu_memory) {

  switch (polynomial_size) {
  case 512:
    scratch_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<512>>(
        v_stream, gpu_index, cbs_vp_buffer, cbs_delta_log, glwe_dimension,
        lwe_dimension, polynomial_size, level_count_cbs, number_of_inputs, tau,
        max_shared_memory, allocate_gpu_memory);
    break;
  case 1024:
    scratch_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<1024>>(
        v_stream, gpu_index, cbs_vp_buffer, cbs_delta_log, glwe_dimension,
        lwe_dimension, polynomial_size, level_count_cbs, number_of_inputs, tau,
        max_shared_memory, allocate_gpu_memory);
    break;
  case 2048:
    scratch_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<2048>>(
        v_stream, gpu_index, cbs_vp_buffer, cbs_delta_log, glwe_dimension,
        lwe_dimension, polynomial_size, level_count_cbs, number_of_inputs, tau,
        max_shared_memory, allocate_gpu_memory);
    break;
  case 4096:
    scratch_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<4096>>(
        v_stream, gpu_index, cbs_vp_buffer, cbs_delta_log, glwe_dimension,
        lwe_dimension, polynomial_size, level_count_cbs, number_of_inputs, tau,
        max_shared_memory, allocate_gpu_memory);
    break;
  case 8192:
    scratch_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<8192>>(
        v_stream, gpu_index, cbs_vp_buffer, cbs_delta_log, glwe_dimension,
        lwe_dimension, polynomial_size, level_count_cbs, number_of_inputs, tau,
        max_shared_memory, allocate_gpu_memory);
    break;
  default:
    break;
  }
}

/*
 * This scratch function allocates the necessary amount of data on the GPU for
 * the wop PBS on 32 bits inputs, into `wop_pbs_buffer`. It also fills the value
 * of delta_log and cbs_delta_log to be used in the bit extract and circuit
 * bootstrap.
 */
void scratch_cuda_wop_pbs_32(
    void *v_stream, uint32_t gpu_index, int8_t **wop_pbs_buffer,
    uint32_t *delta_log, uint32_t *cbs_delta_log, uint32_t glwe_dimension,
    uint32_t lwe_dimension, uint32_t polynomial_size, uint32_t level_count_cbs,
    uint32_t level_count_bsk,
    uint32_t number_of_bits_of_message_including_padding,
    uint32_t number_of_bits_to_extract, uint32_t number_of_inputs,
    uint32_t max_shared_memory, bool allocate_gpu_memory) {
  switch (polynomial_size) {
  case 512:
    scratch_wop_pbs<uint32_t, int32_t, Degree<512>>(
        v_stream, gpu_index, wop_pbs_buffer, delta_log, cbs_delta_log,
        glwe_dimension, lwe_dimension, polynomial_size, level_count_cbs,
        level_count_bsk, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  case 1024:
    scratch_wop_pbs<uint32_t, int32_t, Degree<1024>>(
        v_stream, gpu_index, wop_pbs_buffer, delta_log, cbs_delta_log,
        glwe_dimension, lwe_dimension, polynomial_size, level_count_cbs,
        level_count_bsk, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  case 2048:
    scratch_wop_pbs<uint32_t, int32_t, Degree<2048>>(
        v_stream, gpu_index, wop_pbs_buffer, delta_log, cbs_delta_log,
        glwe_dimension, lwe_dimension, polynomial_size, level_count_cbs,
        level_count_bsk, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  case 4096:
    scratch_wop_pbs<uint32_t, int32_t, Degree<4096>>(
        v_stream, gpu_index, wop_pbs_buffer, delta_log, cbs_delta_log,
        glwe_dimension, lwe_dimension, polynomial_size, level_count_cbs,
        level_count_bsk, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  case 8192:
    scratch_wop_pbs<uint32_t, int32_t, Degree<8192>>(
        v_stream, gpu_index, wop_pbs_buffer, delta_log, cbs_delta_log,
        glwe_dimension, lwe_dimension, polynomial_size, level_count_cbs,
        level_count_bsk, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  default:
    break;
  }
}

/*
 * This scratch function allocates the necessary amount of data on the GPU for
 * the wop PBS on 64 bits inputs, into `wop_pbs_buffer`. It also fills the value
 * of delta_log and cbs_delta_log to be used in the bit extract and circuit
 * bootstrap.
 */
void scratch_cuda_wop_pbs_64(
    void *v_stream, uint32_t gpu_index, int8_t **wop_pbs_buffer,
    uint32_t *delta_log, uint32_t *cbs_delta_log, uint32_t glwe_dimension,
    uint32_t lwe_dimension, uint32_t polynomial_size, uint32_t level_count_cbs,
    uint32_t level_count_bsk,
    uint32_t number_of_bits_of_message_including_padding,
    uint32_t number_of_bits_to_extract, uint32_t number_of_inputs,
    uint32_t max_shared_memory, bool allocate_gpu_memory) {
  switch (polynomial_size) {
  case 512:
    scratch_wop_pbs<uint64_t, int64_t, Degree<512>>(
        v_stream, gpu_index, wop_pbs_buffer, delta_log, cbs_delta_log,
        glwe_dimension, lwe_dimension, polynomial_size, level_count_cbs,
        level_count_bsk, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  case 1024:
    scratch_wop_pbs<uint64_t, int64_t, Degree<1024>>(
        v_stream, gpu_index, wop_pbs_buffer, delta_log, cbs_delta_log,
        glwe_dimension, lwe_dimension, polynomial_size, level_count_cbs,
        level_count_bsk, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  case 2048:
    scratch_wop_pbs<uint64_t, int64_t, Degree<2048>>(
        v_stream, gpu_index, wop_pbs_buffer, delta_log, cbs_delta_log,
        glwe_dimension, lwe_dimension, polynomial_size, level_count_cbs,
        level_count_bsk, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  case 4096:
    scratch_wop_pbs<uint64_t, int64_t, Degree<4096>>(
        v_stream, gpu_index, wop_pbs_buffer, delta_log, cbs_delta_log,
        glwe_dimension, lwe_dimension, polynomial_size, level_count_cbs,
        level_count_bsk, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  case 8192:
    scratch_wop_pbs<uint64_t, int64_t, Degree<8192>>(
        v_stream, gpu_index, wop_pbs_buffer, delta_log, cbs_delta_log,
        glwe_dimension, lwe_dimension, polynomial_size, level_count_cbs,
        level_count_bsk, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, number_of_inputs, max_shared_memory,
        allocate_gpu_memory);
    break;
  default:
    break;
  }
}

/*
 * Entry point for cuda circuit bootstrap + vertical packing for batches of
 * input 64 bit LWE ciphertexts.
 *  - `v_stream` is a void pointer to the Cuda stream to be used in the kernel
 * launch
 *  - `gpu_index` is the index of the GPU to be used in the kernel launch
 *  - 'lwe_array_out' list of output lwe ciphertexts
 *  - 'lwe_array_in' list of input lwe_ciphertexts
 *  - 'fourier_bsk' bootstrapping key in fourier domain, expected half size
 * compressed complex key.
 *  - 'cbs_fpksk' list of private functional packing keyswitch keys
 *  - 'lut_vector' list of test vectors
 *  - 'cbs_vp_buffer' a pre-allocated array to store intermediate results
 *  - 'polynomial_size' size of the test polynomial, supported sizes:
 * {512, 1024, 2048, 4096, 8192}
 *  - 'glwe_dimension' supported dimensions: {1}
 *  - 'lwe_dimension' dimension of input LWE ciphertexts
 *  - 'level_count_bsk' decomposition level for bootstrapping
 *  - 'base_log_bsk'  base log parameter for bootstrapping
 *  - 'level_count_pksk' decomposition level for fp-keyswitch
 *  - 'base_log_pksk' base log parameter for fp-keyswitch
 *  - 'level_count_cbs' level of circuit bootstrap
 *  - 'base_log_cbs' base log parameter for circuit bootstrap
 *  - 'number_of_inputs' number of input LWE ciphertexts
 *  - 'max_shared_memory' maximum shared memory amount to be used in
 *  bootstrapping kernel
 *
 */
void cuda_circuit_bootstrap_vertical_packing_64(
    void *v_stream, uint32_t gpu_index, void *lwe_array_out, void *lwe_array_in,
    void *fourier_bsk, void *cbs_fpksk, void *lut_vector, int8_t *cbs_vp_buffer,
    uint32_t cbs_delta_log, uint32_t polynomial_size, uint32_t glwe_dimension,
    uint32_t lwe_dimension, uint32_t level_count_bsk, uint32_t base_log_bsk,
    uint32_t level_count_pksk, uint32_t base_log_pksk, uint32_t level_count_cbs,
    uint32_t base_log_cbs, uint32_t number_of_inputs, uint32_t lut_number,
    uint32_t max_shared_memory) {
  assert(("Error (GPU circuit bootstrap): polynomial_size should be one of "
          "512, 1024, 2048, 4096, 8192",
          polynomial_size == 512 || polynomial_size == 1024 ||
              polynomial_size == 2048 || polynomial_size == 4096 ||
              polynomial_size == 8192));
  // The number of inputs should be lower than the number of streaming
  // multiprocessors divided by (4 * (k + 1) * l) (the factor 4 being related
  // to the occupancy of 50%). The only supported value for k is 1, so
  // k + 1 = 2 for now.
  int number_of_sm = 0;
  hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, 0);
  assert(("Error (GPU extract bits): the number of input LWEs must be lower or "
          "equal to the "
          "number of streaming multiprocessors on the device divided by 8 * "
          "level_count_bsk",
          number_of_inputs <= number_of_sm / 4. / 2. / level_count_bsk));
  switch (polynomial_size) {
  case 512:
    host_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<512>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)cbs_fpksk, cbs_vp_buffer,
        cbs_delta_log, glwe_dimension, lwe_dimension, polynomial_size,
        base_log_bsk, level_count_bsk, base_log_pksk, level_count_pksk,
        base_log_cbs, level_count_cbs, number_of_inputs, lut_number,
        max_shared_memory);
    break;
  case 1024:
    host_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<1024>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)cbs_fpksk, cbs_vp_buffer,
        cbs_delta_log, glwe_dimension, lwe_dimension, polynomial_size,
        base_log_bsk, level_count_bsk, base_log_pksk, level_count_pksk,
        base_log_cbs, level_count_cbs, number_of_inputs, lut_number,
        max_shared_memory);
    break;
  case 2048:
    host_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<2048>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)cbs_fpksk, cbs_vp_buffer,
        cbs_delta_log, glwe_dimension, lwe_dimension, polynomial_size,
        base_log_bsk, level_count_bsk, base_log_pksk, level_count_pksk,
        base_log_cbs, level_count_cbs, number_of_inputs, lut_number,
        max_shared_memory);
    break;
  case 4096:
    host_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<4096>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)cbs_fpksk, cbs_vp_buffer,
        cbs_delta_log, glwe_dimension, lwe_dimension, polynomial_size,
        base_log_bsk, level_count_bsk, base_log_pksk, level_count_pksk,
        base_log_cbs, level_count_cbs, number_of_inputs, lut_number,
        max_shared_memory);
    break;
  case 8192:
    host_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<8192>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)cbs_fpksk, cbs_vp_buffer,
        cbs_delta_log, glwe_dimension, lwe_dimension, polynomial_size,
        base_log_bsk, level_count_bsk, base_log_pksk, level_count_pksk,
        base_log_cbs, level_count_cbs, number_of_inputs, lut_number,
        max_shared_memory);
    break;
  default:
    break;
  }
}

/*
 * Entry point for entire without padding programmable bootstrap on 64 bit input
 * LWE ciphertexts.
 *  - `v_stream` is a void pointer to the Cuda stream to be used in the kernel
 * launch
 *  - `gpu_index` is the index of the GPU to be used in the kernel launch
 *  - 'lwe_array_out' list of output lwe ciphertexts
 *  - 'lwe_array_in' list of input lwe_ciphertexts
 *  - 'lut_vector' list of test vectors
 *  - 'fourier_bsk' bootstrapping key in fourier domain, expected half size
 * compressed complex key.
 *  - 'ksk' keyswitch key to use inside extract bits block
 *  - 'cbs_fpksk' list of fp-keyswitch keys
 *  - 'wop_pbs_buffer' a pre-allocated array to store intermediate results
 *  - 'glwe_dimension' supported dimensions: {1}
 *  - 'lwe_dimension' dimension of input lwe ciphertexts
 *  - 'polynomial_size' size of the test polynomial, supported sizes:
 * {512, 1024, 2048, 4096, 8192}
 *  - 'base_log_bsk'  base log parameter for bootstrapping
 *  - 'level_count_bsk' decomposition level for bootstrapping
 *  - 'base_log_ksk' base log parameter for keyswitch
 *  - 'level_count_ksk' decomposition level for keyswitch
 *  - 'base_log_pksk' base log parameter for fp-keyswitch
 *  - 'level_count_pksk' decomposition level for fp-keyswitch
 *  - 'base_log_cbs' base log parameter for circuit bootstrap
 *  - 'level_count_cbs' level of circuit bootstrap
 *  - 'number_of_bits_of_message_including_padding' number of bits to extract
 * from each input lwe ciphertext including padding bit
 *  - 'number_of_bits_to_extract' number of bits to extract
 * from each input lwe ciphertext without padding bit
 *  - 'number_of_inputs' number of input lwe ciphertexts
 *  - 'max_shared_memory' maximum shared memory amount to be used in
 *  bootstrapping kernel
 *
 */
void cuda_wop_pbs_64(void *v_stream, uint32_t gpu_index, void *lwe_array_out,
                     void *lwe_array_in, void *lut_vector, void *fourier_bsk,
                     void *ksk, void *cbs_fpksk, int8_t *wop_pbs_buffer,
                     uint32_t cbs_delta_log, uint32_t glwe_dimension,
                     uint32_t lwe_dimension, uint32_t polynomial_size,
                     uint32_t base_log_bsk, uint32_t level_count_bsk,
                     uint32_t base_log_ksk, uint32_t level_count_ksk,
                     uint32_t base_log_pksk, uint32_t level_count_pksk,
                     uint32_t base_log_cbs, uint32_t level_count_cbs,
                     uint32_t number_of_bits_of_message_including_padding,
                     uint32_t number_of_bits_to_extract, uint32_t delta_log,
                     uint32_t number_of_inputs, uint32_t max_shared_memory) {
  assert(("Error (GPU WOP PBS): polynomial_size should be one of "
          "512, 1024, 2048, 4096, 8192",
          polynomial_size == 512 || polynomial_size == 1024 ||
              polynomial_size == 2048 || polynomial_size == 4096 ||
              polynomial_size == 8192));
  // The number of inputs should be lower than the number of streaming
  // multiprocessors divided by (4 * (k + 1) * l) (the factor 4 being related
  // to the occupancy of 50%). The only supported value for k is 1, so
  // k + 1 = 2 for now.
  int number_of_sm = 0;
  hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, 0);
  assert(("Error (GPU WOP PBS): the number of input LWEs must be lower or "
          "equal to the "
          "number of streaming multiprocessors on the device divided by 8 * "
          "level_count_bsk",
          number_of_inputs <= number_of_sm / 4. / 2. / level_count_bsk));
  switch (polynomial_size) {
  case 512:
    host_wop_pbs<uint64_t, int64_t, Degree<512>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)ksk, (uint64_t *)cbs_fpksk,
        wop_pbs_buffer, cbs_delta_log, glwe_dimension, lwe_dimension,
        polynomial_size, base_log_bsk, level_count_bsk, base_log_ksk,
        level_count_ksk, base_log_pksk, level_count_pksk, base_log_cbs,
        level_count_cbs, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, delta_log, number_of_inputs,
        max_shared_memory);
    break;
  case 1024:
    host_wop_pbs<uint64_t, int64_t, Degree<1024>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)ksk, (uint64_t *)cbs_fpksk,
        wop_pbs_buffer, cbs_delta_log, glwe_dimension, lwe_dimension,
        polynomial_size, base_log_bsk, level_count_bsk, base_log_ksk,
        level_count_ksk, base_log_pksk, level_count_pksk, base_log_cbs,
        level_count_cbs, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, delta_log, number_of_inputs,
        max_shared_memory);
    break;
  case 2048:
    host_wop_pbs<uint64_t, int64_t, Degree<2048>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)ksk, (uint64_t *)cbs_fpksk,
        wop_pbs_buffer, cbs_delta_log, glwe_dimension, lwe_dimension,
        polynomial_size, base_log_bsk, level_count_bsk, base_log_ksk,
        level_count_ksk, base_log_pksk, level_count_pksk, base_log_cbs,
        level_count_cbs, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, delta_log, number_of_inputs,
        max_shared_memory);
    break;
  case 4096:
    host_wop_pbs<uint64_t, int64_t, Degree<4096>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)ksk, (uint64_t *)cbs_fpksk,
        wop_pbs_buffer, cbs_delta_log, glwe_dimension, lwe_dimension,
        polynomial_size, base_log_bsk, level_count_bsk, base_log_ksk,
        level_count_ksk, base_log_pksk, level_count_pksk, base_log_cbs,
        level_count_cbs, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, delta_log, number_of_inputs,
        max_shared_memory);
    break;
  case 8192:
    host_wop_pbs<uint64_t, int64_t, Degree<8192>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)ksk, (uint64_t *)cbs_fpksk,
        wop_pbs_buffer, cbs_delta_log, glwe_dimension, lwe_dimension,
        polynomial_size, base_log_bsk, level_count_bsk, base_log_ksk,
        level_count_ksk, base_log_pksk, level_count_pksk, base_log_cbs,
        level_count_cbs, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, delta_log, number_of_inputs,
        max_shared_memory);
    break;
  default:
    break;
  }
}

/*
 * This cleanup function frees the data for the wop PBS on GPU in wop_pbs_buffer
 * for 32 or 64 bits inputs.
 */
void cleanup_cuda_wop_pbs(void *v_stream, uint32_t gpu_index,
                          int8_t **wop_pbs_buffer) {
  auto stream = static_cast<hipStream_t *>(v_stream);
  check_cuda_error(hipStreamSynchronize(*stream));
  // Free memory
  cuda_drop_async(*wop_pbs_buffer, stream, gpu_index);
}

/*
 * This cleanup function frees the data for the circuit bootstrap and vertical
 * packing on GPU in cbs_vp_buffer for 32 or 64 bits inputs.
 */
void cleanup_cuda_circuit_bootstrap_vertical_packing(void *v_stream,
                                                     uint32_t gpu_index,
                                                     int8_t **cbs_vp_buffer) {
  auto stream = static_cast<hipStream_t *>(v_stream);
  check_cuda_error(hipStreamSynchronize(*stream));
  // Free memory
  cuda_drop_async(*cbs_vp_buffer, stream, gpu_index);
}
