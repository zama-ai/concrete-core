#include "hip/hip_runtime.h"
#include "wop_bootstrap.cuh"

void scratch_cuda_wop_pbs_32(
    void *v_stream, uint32_t gpu_index, uint32_t *lut_vector_indexes,
    void *lut_pbs, void *lwe_array_in_buffer, void *lwe_array_in_shifted_buffer,
    void *lwe_array_out_ks_buffer, void *lwe_array_out_pbs_buffer,
    void *lwe_array_out_bit_extract, uint32_t *delta_log,
    uint32_t lwe_dimension, uint32_t polynomial_size,
    uint32_t number_of_bits_of_message_including_padding) {
  scratch_wop_pbs(
      v_stream, gpu_index, lut_vector_indexes, (uint32_t *)lut_pbs,
      (uint32_t *)lwe_array_in_buffer, (uint32_t *)lwe_array_in_shifted_buffer,
      (uint32_t *)lwe_array_out_ks_buffer, (uint32_t *)lwe_array_out_pbs_buffer,
      (uint32_t *)lwe_array_out_bit_extract, delta_log, lwe_dimension,
      polynomial_size, number_of_bits_of_message_including_padding);
}

void scratch_cuda_wop_pbs_64(
    void *v_stream, uint32_t gpu_index, uint32_t *lut_vector_indexes,
    void *lut_pbs, void *lwe_array_in_buffer, void *lwe_array_in_shifted_buffer,
    void *lwe_array_out_ks_buffer, void *lwe_array_out_pbs_buffer,
    void *lwe_array_out_bit_extract, uint32_t *delta_log,
    uint32_t lwe_dimension, uint32_t polynomial_size,
    uint32_t number_of_bits_of_message_including_padding) {
  scratch_wop_pbs(
      v_stream, gpu_index, lut_vector_indexes, (uint64_t *)lut_pbs,
      (uint64_t *)lwe_array_in_buffer, (uint64_t *)lwe_array_in_shifted_buffer,
      (uint64_t *)lwe_array_out_ks_buffer, (uint64_t *)lwe_array_out_pbs_buffer,
      (uint64_t *)lwe_array_out_bit_extract, delta_log, lwe_dimension,
      polynomial_size, number_of_bits_of_message_including_padding);
}

void cuda_circuit_bootstrap_vertical_packing_64(
    void *v_stream, uint32_t gpu_index, void *lwe_array_out, void *lwe_array_in,
    void *fourier_bsk, void *cbs_fpksk, void *lut_vector,
    uint32_t polynomial_size, uint32_t glwe_dimension, uint32_t lwe_dimension,
    uint32_t level_count_bsk, uint32_t base_log_bsk, uint32_t level_count_pksk,
    uint32_t base_log_pksk, uint32_t level_count_cbs, uint32_t base_log_cbs,
    uint32_t number_of_inputs, uint32_t lut_number,
    uint32_t max_shared_memory) {
  assert(("Error (GPU circuit bootstrap): glwe_dimension should be equal to 1",
          glwe_dimension == 1));
  assert(("Error (GPU circuit bootstrap): polynomial_size should be one of "
          "512, 1024, 2048, 4096, 8192",
          polynomial_size == 512 || polynomial_size == 1024 ||
              polynomial_size == 2048 || polynomial_size == 4096 ||
              polynomial_size == 8192));
  // The number of inputs should be lower than the number of streaming
  // multiprocessors divided by (4 * (k + 1) * l) (the factor 4 being related
  // to the occupancy of 50%). The only supported value for k is 1, so
  // k + 1 = 2 for now.
  int number_of_sm = 0;
  hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, 0);
  assert(("Error (GPU extract bits): the number of input LWEs must be lower or "
          "equal to the "
          "number of streaming multiprocessors on the device divided by 8 * "
          "level_count_bsk",
          number_of_inputs <= number_of_sm / 4. / 2. / level_count_bsk));
  switch (polynomial_size) {
  case 512:
    host_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<512>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)cbs_fpksk, glwe_dimension,
        lwe_dimension, polynomial_size, base_log_bsk, level_count_bsk,
        base_log_pksk, level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_inputs, lut_number, max_shared_memory);
    break;
  case 1024:
    host_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<1024>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)cbs_fpksk, glwe_dimension,
        lwe_dimension, polynomial_size, base_log_bsk, level_count_bsk,
        base_log_pksk, level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_inputs, lut_number, max_shared_memory);
    break;
  case 2048:
    host_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<2048>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)cbs_fpksk, glwe_dimension,
        lwe_dimension, polynomial_size, base_log_bsk, level_count_bsk,
        base_log_pksk, level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_inputs, lut_number, max_shared_memory);
    break;
  case 4096:
    host_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<4096>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)cbs_fpksk, glwe_dimension,
        lwe_dimension, polynomial_size, base_log_bsk, level_count_bsk,
        base_log_pksk, level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_inputs, lut_number, max_shared_memory);
    break;
  case 8192:
    host_circuit_bootstrap_vertical_packing<uint64_t, int64_t, Degree<8192>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)cbs_fpksk, glwe_dimension,
        lwe_dimension, polynomial_size, base_log_bsk, level_count_bsk,
        base_log_pksk, level_count_pksk, base_log_cbs, level_count_cbs,
        number_of_inputs, lut_number, max_shared_memory);
    break;
  default:
    break;
  }
}

void cuda_wop_pbs_64(
    void *v_stream, uint32_t gpu_index, void *lwe_array_out, void *lwe_array_in,
    void *lut_vector, void *fourier_bsk, void *ksk, void *cbs_fpksk,
    uint32_t *lut_vector_indexes, void *lut_pbs, void *lwe_array_in_buffer,
    void *lwe_array_in_shifted_buffer, void *lwe_array_out_ks_buffer,
    void *lwe_array_out_pbs_buffer, void *lwe_array_out_bit_extract,
    uint32_t glwe_dimension, uint32_t lwe_dimension, uint32_t polynomial_size,
    uint32_t base_log_bsk, uint32_t level_count_bsk, uint32_t base_log_ksk,
    uint32_t level_count_ksk, uint32_t base_log_pksk, uint32_t level_count_pksk,
    uint32_t base_log_cbs, uint32_t level_count_cbs,
    uint32_t number_of_bits_of_message_including_padding,
    uint32_t number_of_bits_to_extract, uint32_t delta_log,
    uint32_t number_of_inputs, uint32_t max_shared_memory) {
  assert(("Error (GPU WOP PBS): glwe_dimension should be equal to 1",
          glwe_dimension == 1));
  assert(("Error (GPU WOP PBS): polynomial_size should be one of "
          "512, 1024, 2048, 4096, 8192",
          polynomial_size == 512 || polynomial_size == 1024 ||
              polynomial_size == 2048 || polynomial_size == 4096 ||
              polynomial_size == 8192));
  // The number of inputs should be lower than the number of streaming
  // multiprocessors divided by (4 * (k + 1) * l) (the factor 4 being related
  // to the occupancy of 50%). The only supported value for k is 1, so
  // k + 1 = 2 for now.
  int number_of_sm = 0;
  hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, 0);
  assert(("Error (GPU WOP PBS): the number of input LWEs must be lower or "
          "equal to the "
          "number of streaming multiprocessors on the device divided by 8 * "
          "level_count_bsk",
          number_of_inputs <= number_of_sm / 4. / 2. / level_count_bsk));
  switch (polynomial_size) {
  case 512:
    host_wop_pbs<uint64_t, int64_t, Degree<512>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)ksk, (uint64_t *)cbs_fpksk,
        lut_vector_indexes, (uint64_t *)lut_pbs,
        (uint64_t *)lwe_array_in_buffer,
        (uint64_t *)lwe_array_in_shifted_buffer,
        (uint64_t *)lwe_array_out_ks_buffer,
        (uint64_t *)lwe_array_out_pbs_buffer,
        (uint64_t *)lwe_array_out_bit_extract, glwe_dimension, lwe_dimension,
        polynomial_size, base_log_bsk, level_count_bsk, base_log_ksk,
        level_count_ksk, base_log_pksk, level_count_pksk, base_log_cbs,
        level_count_cbs, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, delta_log, number_of_inputs,
        max_shared_memory);
    break;
  case 1024:
    host_wop_pbs<uint64_t, int64_t, Degree<1024>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)ksk, (uint64_t *)cbs_fpksk,
        lut_vector_indexes, (uint64_t *)lut_pbs,
        (uint64_t *)lwe_array_in_buffer,
        (uint64_t *)lwe_array_in_shifted_buffer,
        (uint64_t *)lwe_array_out_ks_buffer,
        (uint64_t *)lwe_array_out_pbs_buffer,
        (uint64_t *)lwe_array_out_bit_extract, glwe_dimension, lwe_dimension,
        polynomial_size, base_log_bsk, level_count_bsk, base_log_ksk,
        level_count_ksk, base_log_pksk, level_count_pksk, base_log_cbs,
        level_count_cbs, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, delta_log, number_of_inputs,
        max_shared_memory);
    break;
  case 2048:
    host_wop_pbs<uint64_t, int64_t, Degree<2048>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)ksk, (uint64_t *)cbs_fpksk,
        lut_vector_indexes, (uint64_t *)lut_pbs,
        (uint64_t *)lwe_array_in_buffer,
        (uint64_t *)lwe_array_in_shifted_buffer,
        (uint64_t *)lwe_array_out_ks_buffer,
        (uint64_t *)lwe_array_out_pbs_buffer,
        (uint64_t *)lwe_array_out_bit_extract, glwe_dimension, lwe_dimension,
        polynomial_size, base_log_bsk, level_count_bsk, base_log_ksk,
        level_count_ksk, base_log_pksk, level_count_pksk, base_log_cbs,
        level_count_cbs, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, delta_log, number_of_inputs,
        max_shared_memory);
    break;
  case 4096:
    host_wop_pbs<uint64_t, int64_t, Degree<4096>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)ksk, (uint64_t *)cbs_fpksk,
        lut_vector_indexes, (uint64_t *)lut_pbs,
        (uint64_t *)lwe_array_in_buffer,
        (uint64_t *)lwe_array_in_shifted_buffer,
        (uint64_t *)lwe_array_out_ks_buffer,
        (uint64_t *)lwe_array_out_pbs_buffer,
        (uint64_t *)lwe_array_out_bit_extract, glwe_dimension, lwe_dimension,
        polynomial_size, base_log_bsk, level_count_bsk, base_log_ksk,
        level_count_ksk, base_log_pksk, level_count_pksk, base_log_cbs,
        level_count_cbs, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, delta_log, number_of_inputs,
        max_shared_memory);
    break;
  case 8192:
    host_wop_pbs<uint64_t, int64_t, Degree<8192>>(
        v_stream, gpu_index, (uint64_t *)lwe_array_out,
        (uint64_t *)lwe_array_in, (uint64_t *)lut_vector,
        (double2 *)fourier_bsk, (uint64_t *)ksk, (uint64_t *)cbs_fpksk,
        lut_vector_indexes, (uint64_t *)lut_pbs,
        (uint64_t *)lwe_array_in_buffer,
        (uint64_t *)lwe_array_in_shifted_buffer,
        (uint64_t *)lwe_array_out_ks_buffer,
        (uint64_t *)lwe_array_out_pbs_buffer,
        (uint64_t *)lwe_array_out_bit_extract, glwe_dimension, lwe_dimension,
        polynomial_size, base_log_bsk, level_count_bsk, base_log_ksk,
        level_count_ksk, base_log_pksk, level_count_pksk, base_log_cbs,
        level_count_cbs, number_of_bits_of_message_including_padding,
        number_of_bits_to_extract, delta_log, number_of_inputs,
        max_shared_memory);
    break;
  default:
    break;
  }
}

void cleanup_cuda_wop_pbs_32(void *v_stream, uint32_t gpu_index,
                             uint32_t *lut_vector_indexes, void *lut_pbs,
                             void *lwe_array_in_buffer,
                             void *lwe_array_in_shifted_buffer,
                             void *lwe_array_out_ks_buffer,
                             void *lwe_array_out_pbs_buffer,
                             void *lwe_array_out_bit_extract) {
  cleanup_wop_pbs(
      v_stream, gpu_index, lut_vector_indexes, (uint32_t *)lut_pbs,
      (uint32_t *)lwe_array_in_buffer, (uint32_t *)lwe_array_in_shifted_buffer,
      (uint32_t *)lwe_array_out_ks_buffer, (uint32_t *)lwe_array_out_pbs_buffer,
      (uint32_t *)lwe_array_out_bit_extract);
}

void cleanup_cuda_wop_pbs_64(void *v_stream, uint32_t gpu_index,
                             uint32_t *lut_vector_indexes, void *lut_pbs,
                             void *lwe_array_in_buffer,
                             void *lwe_array_in_shifted_buffer,
                             void *lwe_array_out_ks_buffer,
                             void *lwe_array_out_pbs_buffer,
                             void *lwe_array_out_bit_extract) {
  cleanup_wop_pbs(
      v_stream, gpu_index, lut_vector_indexes, (uint64_t *)lut_pbs,
      (uint64_t *)lwe_array_in_buffer, (uint64_t *)lwe_array_in_shifted_buffer,
      (uint64_t *)lwe_array_out_ks_buffer, (uint64_t *)lwe_array_out_pbs_buffer,
      (uint64_t *)lwe_array_out_bit_extract);
}
