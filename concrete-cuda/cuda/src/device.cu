#include "device.h"
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

/// Unsafe function to create a CUDA stream, must check first that GPU exists
hipStream_t *cuda_create_stream(uint32_t gpu_index) {
  hipSetDevice(gpu_index);
  hipStream_t *stream = new hipStream_t;
  hipStreamCreate(stream);
  return stream;
}

/// Unsafe function to destroy CUDA stream, must check first the GPU exists
int cuda_destroy_stream(hipStream_t *stream, uint32_t gpu_index) {
  hipSetDevice(gpu_index);
  hipStreamDestroy(*stream);
  return 0;
}

/// Unsafe function that will try to allocate even if gpu_index is invalid
/// or if there's not enough memory. A safe wrapper around it must call
/// cuda_check_valid_malloc() first
void *cuda_malloc(uint64_t size, uint32_t gpu_index) {
  hipSetDevice(gpu_index);
  void *ptr;
  checkCudaErrors(hipMalloc((void **)&ptr, size));

  return ptr;
}

/// Allocates a size-byte array at the device memory. Tries to do it
/// asynchronously.
void *cuda_malloc_async(uint64_t size, hipStream_t *stream,
                        uint32_t gpu_index) {
  void *ptr;

  int support_async_alloc;
  checkCudaErrors(hipDeviceGetAttribute(
      &support_async_alloc, hipDeviceAttributeMemoryPoolsSupported, gpu_index));

  if (support_async_alloc)
    checkCudaErrors(hipMallocAsync((void **)&ptr, size, *stream));
  else
    checkCudaErrors(hipMalloc((void **)&ptr, size));
  return ptr;
}

/// Checks that allocation is valid
/// 0: valid
/// -1: invalid, not enough memory in device
/// -2: invalid, gpu index doesn't exist
int cuda_check_valid_malloc(uint64_t size, uint32_t gpu_index) {

  if (gpu_index >= cuda_get_number_of_gpus()) {
    // error code: invalid gpu_index
    return -2;
  }
  hipSetDevice(gpu_index);
  size_t total_mem, free_mem;
  hipMemGetInfo(&free_mem, &total_mem);
  if (size > free_mem) {
    // error code: not enough memory
    return -1;
  }
  return 0;
}

/// Tries to copy memory to the GPU asynchronously
/// 0: success
/// -1: error, invalid device pointer
/// -2: error, gpu index doesn't exist
/// -3: error, zero copy size
int cuda_memcpy_async_to_gpu(void *dest, void *src, uint64_t size,
                             hipStream_t *stream, uint32_t gpu_index) {
  if (size == 0) {
    // error code: zero copy size
    return -3;
  }

  if (gpu_index >= cuda_get_number_of_gpus()) {
    // error code: invalid gpu_index
    return -2;
  }
  hipPointerAttribute_t attr;
  hipPointerGetAttributes(&attr, dest);
  if (attr.device != gpu_index && attr.type != hipMemoryTypeDevice) {
    // error code: invalid device pointer
    return -1;
  }

  hipSetDevice(gpu_index);
  checkCudaErrors(
      hipMemcpyAsync(dest, src, size, hipMemcpyHostToDevice, *stream));
  return 0;
}

/// Synchronizes device
/// 0: success
/// -2: error, gpu index doesn't exist
int cuda_synchronize_device(uint32_t gpu_index) {
  if (gpu_index >= cuda_get_number_of_gpus()) {
    // error code: invalid gpu_index
    return -2;
  }
  hipSetDevice(gpu_index);
  hipDeviceSynchronize();
  return 0;
}

/// Tries to copy memory to the GPU asynchronously
/// 0: success
/// -1: error, invalid device pointer
/// -2: error, gpu index doesn't exist
/// -3: error, zero copy size
int cuda_memcpy_async_to_cpu(void *dest, const void *src, uint64_t size,
                             hipStream_t *stream, uint32_t gpu_index) {
  if (size == 0) {
    // error code: zero copy size
    return -3;
  }

  if (gpu_index >= cuda_get_number_of_gpus()) {
    // error code: invalid gpu_index
    return -2;
  }
  hipPointerAttribute_t attr;
  hipPointerGetAttributes(&attr, src);
  if (attr.device != gpu_index && attr.type != hipMemoryTypeDevice) {
    // error code: invalid device pointer
    return -1;
  }

  hipSetDevice(gpu_index);
  checkCudaErrors(
      hipMemcpyAsync(dest, src, size, hipMemcpyDeviceToHost, *stream));
  return 0;
}

/// Return number of GPUs available
int cuda_get_number_of_gpus() {
  int num_gpus;
  hipGetDeviceCount(&num_gpus);
  return num_gpus;
}

/// Drop a cuda array
int cuda_drop(void *ptr, uint32_t gpu_index) {
  if (gpu_index >= cuda_get_number_of_gpus()) {
    // error code: invalid gpu_index
    return -2;
  }
  hipSetDevice(gpu_index);
  checkCudaErrors(hipFree(ptr));
  return 0;
}

/// Drop a cuda array. Tries to do it asynchronously
int cuda_drop_async(void *ptr, hipStream_t *stream, uint32_t gpu_index) {

  int support_async_alloc;
  checkCudaErrors(hipDeviceGetAttribute(
      &support_async_alloc, hipDeviceAttributeMemoryPoolsSupported, gpu_index));

  if (support_async_alloc)
    checkCudaErrors(hipFreeAsync(ptr, *stream));
  else
    checkCudaErrors(hipFree(ptr));
  return 0;
}

/// Get the maximum size for the shared memory
int cuda_get_max_shared_memory(uint32_t gpu_index) {
  if (gpu_index >= cuda_get_number_of_gpus()) {
    // error code: invalid gpu_index
    return -2;
  }
  hipSetDevice(gpu_index);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, gpu_index);
  int max_shared_memory = 0;
  if (prop.major > 7) {
    max_shared_memory = prop.sharedMemPerMultiprocessor;
  } else {
    max_shared_memory = prop.sharedMemPerBlock;
  }
  return max_shared_memory;
}

int cuda_synchronize_stream(void *v_stream) {
  auto stream = static_cast<hipStream_t *>(v_stream);
  hipStreamSynchronize(*stream);
  return 0;
}
