#include "hip/hip_runtime.h"
#include "bootstrap_amortized.cuh"

/* Perform bootstrapping on a batch of input LWE ciphertexts
 *
 *  - lwe_array_out: output batch of num_samples bootstrapped ciphertexts c =
 * (a0,..an-1,b) where n is the LWE dimension
 *  - lut_vector: should hold as many test vectors of size polynomial_size
 * as there are input ciphertexts, but actually holds
 * num_lut_vectors vectors to reduce memory usage
 *  - lut_vector_indexes: stores the index corresponding to
 * which test vector to use for each sample in
 * lut_vector
 *  - lwe_array_in: input batch of num_samples LWE ciphertexts, containing n
 * mask values + 1 body value
 *  - bootstrapping_key: RGSW encryption of the LWE secret key sk1
 * under secret key sk2
 * bsk = Z + sk1 H
 * where H is the gadget matrix and Z is a matrix (k+1).l
 * containing GLWE encryptions of 0 under sk2.
 * bsk is thus a tensor of size (k+1)^2.l.N.n
 * where l is the number of decomposition levels and
 * k is the GLWE dimension, N is the polynomial size for
 * GLWE. The polynomial size for GLWE and the test vector
 * are the same because they have to be in the same ring
 * to be multiplied.
 * Note: it is necessary to generate (k+1).k.l.N.n
 * uniformly random coefficients for the zero encryptions
 * - input_lwe_dimension: size of the Torus vector used to encrypt the input
 * LWE ciphertexts - referred to as n above (~ 600)
 * - polynomial_size: size of the test polynomial (test vector) and size of the
 * GLWE polynomial (~1024)
 * - base_log: log base used for the gadget matrix - B = 2^base_log (~8)
 * - level_count: number of decomposition levels in the gadget matrix (~4)
 * - num_samples: number of encrypted input messages
 * - num_lut_vectors: parameter to set the actual number of test vectors to be
 * used
 * - q: number of bytes in the integer representation (32 or 64)
 *
 * This function calls a wrapper to a device kernel that performs the
 * bootstrapping:
 * 	- the kernel is templatized based on integer discretization and
 * polynomial degree
 * 	- num_samples blocks of threads are launched, where each thread is going
 * to handle one or more polynomial coefficients at each stage:
 * 		- perform the blind rotation
 * 		- round the result
 * 		- decompose into level_count levels, then for each level:
 * 		  - switch to the FFT domain
 * 		  - multiply with the bootstrapping key
 * 		  - come back to the coefficients representation
 * 	- between each stage a synchronization of the threads is necessary
 * 	- in case the device has enough shared memory, temporary arrays used for
 * the different stages (accumulators) are stored into the shared memory
 * 	- the accumulators serve to combine the results for all decomposition
 * levels
 * 	- the constant memory (64K) is used for storing the roots of identity
 * values for the FFT
 */

void cuda_bootstrap_amortized_lwe_ciphertext_vector_32(
    void *v_stream, void *lwe_array_out, void *lut_vector,
    void *lut_vector_indexes, void *lwe_array_in, void *bootstrapping_key,
    uint32_t lwe_dimension, uint32_t glwe_dimension, uint32_t polynomial_size,
    uint32_t base_log, uint32_t level_count, uint32_t num_samples,
    uint32_t num_lut_vectors, uint32_t lwe_idx, uint32_t max_shared_memory) {

  assert(
      ("Error (GPU amortized PBS): base log should be <= 8", base_log <= 8));
  assert(("Error (GPU amortized PBS): glwe_dimension should be equal to 1",
          glwe_dimension == 1));
  assert(("Error (GPU amortized PBS): polynomial size should be one of 512, "
          "1024, 2048, 4096, 8192",
          polynomial_size == 512 || polynomial_size == 1024 ||
              polynomial_size == 2048 || polynomial_size == 4096 ||
              polynomial_size == 8192));

  switch (polynomial_size) {
  case 512:
    host_bootstrap_amortized<uint32_t, Degree<512>>(
        v_stream, (uint32_t *)lwe_array_out, (uint32_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint32_t *)lwe_array_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size, base_log,
        level_count, num_samples, num_lut_vectors, lwe_idx, max_shared_memory);
    break;
  case 1024:
    host_bootstrap_amortized<uint32_t, Degree<1024>>(
        v_stream, (uint32_t *)lwe_array_out, (uint32_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint32_t *)lwe_array_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size, base_log,
        level_count, num_samples, num_lut_vectors, lwe_idx, max_shared_memory);
    break;
  case 2048:
    host_bootstrap_amortized<uint32_t, Degree<2048>>(
        v_stream, (uint32_t *)lwe_array_out, (uint32_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint32_t *)lwe_array_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size, base_log,
        level_count, num_samples, num_lut_vectors, lwe_idx, max_shared_memory);
    break;
  case 4096:
    host_bootstrap_amortized<uint32_t, Degree<4096>>(
        v_stream, (uint32_t *)lwe_array_out, (uint32_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint32_t *)lwe_array_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size, base_log,
        level_count, num_samples, num_lut_vectors, lwe_idx, max_shared_memory);
    break;
  case 8192:
    host_bootstrap_amortized<uint32_t, Degree<8192>>(
        v_stream, (uint32_t *)lwe_array_out, (uint32_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint32_t *)lwe_array_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size, base_log,
        level_count, num_samples, num_lut_vectors, lwe_idx, max_shared_memory);
    break;
  default:
    break;
  }
}

void cuda_bootstrap_amortized_lwe_ciphertext_vector_64(
    void *v_stream, void *lwe_array_out, void *lut_vector,
    void *lut_vector_indexes, void *lwe_array_in, void *bootstrapping_key,
    uint32_t lwe_dimension, uint32_t glwe_dimension, uint32_t polynomial_size,
    uint32_t base_log, uint32_t level_count, uint32_t num_samples,
    uint32_t num_lut_vectors, uint32_t lwe_idx, uint32_t max_shared_memory) {

  assert(
      ("Error (GPU amortized PBS): base log should be <= 8", base_log <= 8));
  assert(("Error (GPU amortized PBS): glwe_dimension should be equal to 1",
          glwe_dimension == 1));
  assert(("Error (GPU amortized PBS): polynomial size should be one of 512, "
          "1024, 2048, 4096, 8192",
          polynomial_size == 512 || polynomial_size == 1024 ||
              polynomial_size == 2048 || polynomial_size == 4096 ||
              polynomial_size == 8192));

  switch (polynomial_size) {
  case 512:
    host_bootstrap_amortized<uint64_t, Degree<512>>(
        v_stream, (uint64_t *)lwe_array_out, (uint64_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint64_t *)lwe_array_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size, base_log,
        level_count, num_samples, num_lut_vectors, lwe_idx, max_shared_memory);
    break;
  case 1024:
    host_bootstrap_amortized<uint64_t, Degree<1024>>(
        v_stream, (uint64_t *)lwe_array_out, (uint64_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint64_t *)lwe_array_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size, base_log,
        level_count, num_samples, num_lut_vectors, lwe_idx, max_shared_memory);
    break;
  case 2048:
    host_bootstrap_amortized<uint64_t, Degree<2048>>(
        v_stream, (uint64_t *)lwe_array_out, (uint64_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint64_t *)lwe_array_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size, base_log,
        level_count, num_samples, num_lut_vectors, lwe_idx, max_shared_memory);
    break;
  case 4096:
    host_bootstrap_amortized<uint64_t, Degree<4096>>(
        v_stream, (uint64_t *)lwe_array_out, (uint64_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint64_t *)lwe_array_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size, base_log,
        level_count, num_samples, num_lut_vectors, lwe_idx, max_shared_memory);
    break;
  case 8192:
    host_bootstrap_amortized<uint64_t, Degree<8192>>(
        v_stream, (uint64_t *)lwe_array_out, (uint64_t *)lut_vector,
        (uint32_t *)lut_vector_indexes, (uint64_t *)lwe_array_in,
        (double2 *)bootstrapping_key, lwe_dimension, polynomial_size, base_log,
        level_count, num_samples, num_lut_vectors, lwe_idx, max_shared_memory);
    break;
  default:
    break;
  }
}
